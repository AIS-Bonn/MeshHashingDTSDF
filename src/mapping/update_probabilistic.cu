#include "hip/hip_runtime.h"
//
// Created by wei on 17-10-25.
//

#include <>
#include "update_probabilistic.h"

#include "core/block_array.h"
#include "mapping/update_simple.h"
#include "engine/main_engine.h"
#include "sensor/rgbd_sensor.h"
#include "geometry/spatial_query.h"

////////////////////
/// Device code
////////////////////
__global__
void RefineSensorDataKernel(
    EntryArray candidate_entries,
    BlockArray blocks,
    Mesh mesh,
    SensorData sensor_data,
    SensorParams sensor_params,
    float4x4 c_T_w,
    HashTable hash_table,
    GeometryHelper geometry_helper) {

  //TODO check if we should load this in shared memory (entries)
  /// 1. Select voxel
  const HashEntry &entry = candidate_entries[blockIdx.x];
  int3 voxel_base_pos = geometry_helper.BlockToVoxel(entry.pos);
  uint local_idx = threadIdx.x;  //inside of an SDF block
  int3 voxel_pos = voxel_base_pos + make_int3(geometry_helper.IdxToVoxelLocalPos(local_idx));

  Voxel &this_voxel = blocks[entry.ptr].voxels[local_idx];
  /// 2. Project to camera
  float3 world_pos = geometry_helper.VoxelToWorld(voxel_pos);
  float3 camera_pos = c_T_w * world_pos;
  uint2 image_pos = make_uint2(
      geometry_helper.CameraProjectToImagei(camera_pos,
                                            sensor_params.fx, sensor_params.fy,
                                            sensor_params.cx, sensor_params.cy));
  if (image_pos.x >= sensor_params.width
      || image_pos.y >= sensor_params.height)
    return;

  /// 3. Find correspondent depth observation
  float depth = tex2D<float>(sensor_data.depth_texture, image_pos.x, image_pos.y);
  if (depth == MINF || depth == 0.0f || depth >= geometry_helper.sdf_upper_bound)
    return;

  /// 4. SDF computation
  float3 dp = geometry_helper.ImageReprojectToCamera(image_pos.x, image_pos.y, depth,
                                                     sensor_params.fx, sensor_params.fy, sensor_params.cx,
                                                     sensor_params.cy);
  float3 dpw = c_T_w.getInverse() * dp;

  /// Solve (I + \sum \lambda nn^T + ... )x = (dp + \sum \lambda nn^Tv)
  float3x3 A = float3x3::getIdentity();
  float3 b = dpw;
  float wd = (1.0f - geometry_helper.NormalizeDepth(depth,
                                                    sensor_params.min_depth_range,
                                                    sensor_params.max_depth_range));
  float wn = 0.5f;
  bool addition = false;
  for (int i = 0; i < N_VERTEX; ++i) {
    if (this_voxel.vertex_ptrs[i] > 0) {
      addition = true;
      Vertex vtx = mesh.vertex(this_voxel.vertex_ptrs[i]);
      float3 v = vtx.pos;
      float3 n = vtx.normal;
      wn += dot(c_T_w * n, normalize(-dp));
      float3x3 nnT = float3x3(n.x * n.x, n.x * n.y, n.x * n.z,
                              n.y * n.x, n.y * n.y, n.y * n.z,
                              n.z * n.x, n.z * n.y, n.z * n.z);

      float dist = length(dpw - v);
      float wdist = dist / geometry_helper.voxel_size;
      float ww = expf(-wdist * wdist);
      A = A + nnT * ww;
      b = b + nnT * v * ww;
    }
  }

  // Best estimation for dp
  if (addition) {
    dpw = A.getInverse() * b;
  }
  dp = c_T_w * dpw;
  //float3 np = normalize(-dp);

  //printf("%f %f %f\n", np.x, np.y, np.z)

  //float sdf = dot(normalize(-dp), camera_pos - dp);
  float sdf = depth - camera_pos.z;
  //uchar weight = (uchar)fmax(1.0f, kVolumeParams.weight_sample * wn * wd);

  float weight = (uchar) fmax(geometry_helper.weight_sample * 1.5f *
                              (1.0f - geometry_helper.NormalizeDepth(depth,
                                                                     sensor_params.min_depth_range,
                                                                     sensor_params.max_depth_range)),
                              1.0f);
  float truncation = geometry_helper.truncate_distance(depth);
  if (sdf <= -truncation)
    return;
  if (sdf >= 0.0f) {
    sdf = fminf(truncation, sdf);
  } else {
    sdf = fmaxf(-truncation, sdf);
  }

  /// 5. Update
  Voxel delta;
  delta.sdf = sdf;
  delta.weight = weight;

  if (sensor_data.color_data) {
    float4 color = tex2D<float4>(sensor_data.color_texture, image_pos.x, image_pos.y);
    delta.color = make_uchar3(255 * color.x, 255 * color.y, 255 * color.z);
  } else {
    delta.color = make_uchar3(0, 255, 0);
  }
  this_voxel.Update(delta);
}

void RefineSensorData(EntryArray &candidate_entries,
                      BlockArray &blocks,
                      Mesh &mesh,
                      Sensor &sensor,
                      HashTable &hash_table,
                      GeometryHelper &geometry_helper) {
  const uint threads_per_block = BLOCK_SIZE;

  uint compacted_entry_count = candidate_entries.count();
  if (compacted_entry_count <= 0)
    return;

  const dim3 grid_size(compacted_entry_count, 1);
  const dim3 block_size(threads_per_block, 1);
  RefineSensorDataKernel << < grid_size, block_size >> > (
      candidate_entries,
          blocks,
          mesh,
          sensor.data(),
          sensor.sensor_params(),
          sensor.cTw(),
          hash_table,
          geometry_helper);
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipGetLastError());
}