#include "hip/hip_runtime.h"
//
// Created by wei on 17-10-22.
//

#include "mapping/recycle.h"

////////////////////
/// Device code
////////////////////
#include "core/common.h"
#include "core/entry_array.h"
#include "core/block_array.h"
#include "hip/hip_vector_types.h"

__global__
void StarveOccupiedBlocksKernel(
    EntryArray candidate_entries,
    BlockArray blocks
) {
  const uint idx = blockIdx.x;
  const HashEntry& entry = candidate_entries[idx];
  float weight = blocks[entry.ptr].voxels[threadIdx.x].weight;
  weight = fmaxf(0, weight - 1.0f);
  blocks[entry.ptr].voxels[threadIdx.x].weight = weight;
}

/// Collect dead voxels
__global__
void CollectGarbageBlockArrayKernel(
    EntryArray candidate_entries,
    BlockArray blocks,
    GeometryHelper geometry_helper
) {

  const uint idx = blockIdx.x;
  const HashEntry& entry = candidate_entries[idx];

  Voxel v0 = blocks[entry.ptr].voxels[2*threadIdx.x+0];
  Voxel v1 = blocks[entry.ptr].voxels[2*threadIdx.x+1];

  float sdf0 = v0.sdf, sdf1 = v1.sdf;
  if (v0.weight < EPSILON)	sdf0 = PINF;
  if (v1.weight < EPSILON)	sdf1 = PINF;

  __shared__ float	shared_min_sdf   [BLOCK_SIZE / 2];
  __shared__ float	shared_max_weight[BLOCK_SIZE / 2];
  shared_min_sdf[threadIdx.x] = fminf(fabsf(sdf0), fabsf(sdf1));
  shared_max_weight[threadIdx.x] = fmaxf(v0.weight, v1.weight);

  /// reducing operation
#pragma unroll 1
  for (uint stride = 2; stride <= blockDim.x; stride <<= 1) {

    __syncthreads();
    if ((threadIdx.x  & (stride-1)) == (stride-1)) {
      shared_min_sdf[threadIdx.x] = fminf(shared_min_sdf[threadIdx.x-stride/2],
                                          shared_min_sdf[threadIdx.x]);
      shared_max_weight[threadIdx.x] = fmaxf(shared_max_weight[threadIdx.x-stride/2],
                                             shared_max_weight[threadIdx.x]);
    }
  }
  __syncthreads();

  if (threadIdx.x == blockDim.x - 1) {
    float min_sdf = shared_min_sdf[threadIdx.x];
    float max_weight = shared_max_weight[threadIdx.x];

    // TODO(wei): check this weird reference
    float t = geometry_helper.truncate_distance(5.0f);

    // TODO(wei): add || valid_triangles == 0 when memory leak is dealt with
    candidate_entries.flag(idx) =
        (min_sdf >= t || max_weight < EPSILON) ? (uchar)1 : (uchar)0;
  }
}

/// !!! Their mesh not recycled
__global__
void RecycleGarbageTrianglesKernel(
    EntryArray candidate_entries,
    BlockArray blocks,
    Mesh       mesh,
    HashTable  hash_table
) {
  const uint idx = blockIdx.x;
  if (candidate_entries.flag(idx) == 0) return;

  const HashEntry& entry = candidate_entries[idx];
  const uint local_idx = threadIdx.x;  //inside an SDF block
  Voxel &voxel = blocks[entry.ptr].voxels[local_idx];

  for (int i = 0; i < N_TRIANGLE; ++i) {
    int triangle_ptr = voxel.triangle_ptrs[i];
    if (triangle_ptr == FREE_PTR) continue;

    // Clear ref_count of its pointed vertices
    mesh.ReleaseTriangle(mesh.triangle(triangle_ptr));
    mesh.triangle(triangle_ptr).Clear();
    mesh.FreeTriangle(triangle_ptr);
    voxel.triangle_ptrs[i] = FREE_PTR;
  }
}

__global__
void RecycleGarbageVerticesKernel(
    EntryArray candidate_entries,
    BlockArray       blocks,
    Mesh             mesh,
    HashTable        hash_table
) {
  if (candidate_entries.flag(blockIdx.x) == 0) return;
  const HashEntry &entry = candidate_entries[blockIdx.x];
  const uint local_idx = threadIdx.x;

  Voxel &cube = blocks[entry.ptr].voxels[local_idx];

  __shared__ int valid_vertex_count;
  if (threadIdx.x == 0) valid_vertex_count = 0;
  __syncthreads();

#pragma unroll 1
  for (int i = 0; i < 3; ++i) {
    if (cube.vertex_ptrs[i] != FREE_PTR) {
      if (mesh.vertex(cube.vertex_ptrs[i]).ref_count <= 0) {
        mesh.vertex(cube.vertex_ptrs[i]).Clear();
        mesh.FreeVertex(cube.vertex_ptrs[i]);
        cube.vertex_ptrs[i] = FREE_PTR;
      }
      else {
        atomicAdd(&valid_vertex_count, 1);
      }
    }
  }

  __syncthreads();
  if (threadIdx.x == 0 && valid_vertex_count == 0) {
    if (hash_table.FreeEntry(entry.pos)) {
      blocks[entry.ptr].Clear();
    }
  }
}

void StarveOccupiedBlockArray(
    EntryArray& candidate_entries,
    BlockArray& blocks
) {
  const uint threads_per_block = BLOCK_SIZE;

  uint processing_block_count = candidate_entries.count();
  if (processing_block_count <= 0)
    return;

  const dim3 grid_size(processing_block_count, 1);
  const dim3 block_size(threads_per_block, 1);

  StarveOccupiedBlocksKernel<<<grid_size, block_size >>>(candidate_entries, blocks);
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipGetLastError());
}

void CollectGarbageBlockArray(
    EntryArray& candidate_entries,
    BlockArray& blocks,
    GeometryHelper& geometry_helper
) {
  const uint threads_per_block = BLOCK_SIZE / 2;

  uint processing_block_count = candidate_entries.count();
  if (processing_block_count <= 0)
    return;

  const dim3 grid_size(processing_block_count, 1);
  const dim3 block_size(threads_per_block, 1);

  CollectGarbageBlockArrayKernel <<<grid_size, block_size >>>(
      candidate_entries,
          blocks,
          geometry_helper);
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipGetLastError());
}

// TODO(wei): Check vertex / triangles in detail
// including garbage collection
void RecycleGarbageBlockArray(
    EntryArray &candidate_entries,
    BlockArray& blocks,
    Mesh&      mesh,
    HashTable& hash_table
) {
  const uint threads_per_block = BLOCK_SIZE;

  uint processing_block_count = candidate_entries.count();
  if (processing_block_count <= 0)
    return;

  const dim3 grid_size(processing_block_count, 1);
  const dim3 block_size(threads_per_block, 1);

  RecycleGarbageTrianglesKernel <<<grid_size, block_size >>>(
      candidate_entries, blocks, mesh, hash_table);
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipGetLastError());

  RecycleGarbageVerticesKernel <<<grid_size, block_size >>>(
      candidate_entries, blocks, mesh, hash_table);
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipGetLastError());
}

