#include "hip/hip_runtime.h"
#include "core/directional_tsdf.h"
#include "core/functions.h"
#include "mapping/allocate.h"
#include "mapping/update_raycasting.h"
#include "mapping/block_traversal.hpp"
#include "util/timer.h"
#include "geometry/geometry_helper.h"

////////////////////
/// Device code
////////////////////

__device__
inline void UpdateVoxel(
    const int3 &voxel_idx,
    const size_t voxel_array_idx,
    const float normalized_depth,
    const float truncation_distance,
    const float3 &surface_point_world,
    const float3 &normal_world,
    const float4 &normal_camera,
    const BlockArray &blocks,
    const SensorParams &sensor_params,
    const HashTable &hash_table,
    GeometryHelper &geometry_helper,
    const bool enable_point_to_plane
)
{
  int3 block_idx = geometry_helper.VoxelToBlock(voxel_idx);
  uint local_idx = geometry_helper.VectorizeOffset(geometry_helper.VoxelToOffset(block_idx, voxel_idx));

  if (not blocks.HasVoxelArray(hash_table.GetEntry(block_idx).ptr, voxel_array_idx))
  {
    printf("(%i, %i, %i) ", voxel_idx.x, voxel_idx.y, voxel_idx.z);
    return;
  }
  Voxel &voxel = blocks.GetVoxelArray(hash_table.GetEntry(block_idx).ptr, voxel_array_idx).voxels[local_idx];

  float3 voxel_pos_world = geometry_helper.VoxelToWorld(voxel_idx);

//    float weight = fmaxf(10 * geometry_helper.weight_sample * (1.0f - normalized_depth), 1.0f);

//    // linear voxel-observation-distance weight
//    float weight = fmaxf(10 * geometry_helper.weight_sample * (1.0f - normalized_depth) *
//                         length(point_world_pos - voxel_pos_world) / truncation_distance, 1.0f);
  // linear voxel-observation-distance weight + normal angle
  float weight = fmaxf(10 * geometry_helper.weight_sample * (1.0f - normalized_depth) *
                       length(surface_point_world - voxel_pos_world) / truncation_distance *
                       (2 - normal_camera.x + normal_camera.y),
                       1.0f);

  float3 observation_ray = voxel_pos_world - surface_point_world;
  float sdf;
  if (enable_point_to_plane)
    sdf = dot(observation_ray, normal_world);
  else
    sdf = sign(dot(observation_ray, normal_world)) * length(observation_ray);

  atomicAdd(&voxel.a, weight * sdf);
  atomicAdd(&voxel.b, weight);
  atomicAdd(&voxel.num_updates, 1);
}

/**
 * For every pixel casts a ray into multiple voxels (truncation range and updates the
 * SDF and weight summation values)
 * @param blocks
 * @param sensor_data
 * @param sensor_params
 * @param runtime_params
 * @param wTc
 * @param hash_table
 * @param geometry_helper
 */
__global__
void UpdateRaycastingKernel(
    BlockArray blocks,
    SensorData sensor_data,
    SensorParams sensor_params,
    RuntimeParams runtime_params,
    float4x4 wTc,
    HashTable hash_table,
    GeometryHelper geometry_helper
)
{
  // Pixel coordinates
  const int ux = blockIdx.x * blockDim.x + threadIdx.x;
  const int uy = blockIdx.y * blockDim.y + threadIdx.y;

  if (ux >= sensor_params.width || uy >= sensor_params.height)
    return;

  float depth = tex2D<float>(sensor_data.depth_texture, ux, uy);
  float4 normal_camera = tex2D<float4>(sensor_data.normal_texture, ux, uy);

  if (not IsValidDepth(depth) or depth >= geometry_helper.sdf_upper_bound or not IsValidNormal(normal_camera))
    return;

  float3 point_camera_pos = GeometryHelper::ImageReprojectToCamera(ux, uy, depth,
                                                                   sensor_params.fx, sensor_params.fy,
                                                                   sensor_params.cx, sensor_params.cy);
  float3 point_world_pos = make_float3(wTc * make_float4(point_camera_pos, 1));

  float4x4 wTcRotOnly = wTc;
  wTcRotOnly.m14 = 0;
  wTcRotOnly.m24 = 0;
  wTcRotOnly.m34 = 0;
  float3 normal_world = make_float3(wTcRotOnly * normal_camera);

  float truncation_distance = geometry_helper.truncate_distance(depth);

  // Traverse voxels in normal's direction through measured surface point
  float3 ray_origin;
  float3 ray_direction;

  if (runtime_params.raycasting_mode == RAY_DIRECTION_CAMERA)
  {
    float3 camera_world_pos = make_float3(wTc * make_float4(0, 0, 0, 1));
    ray_direction = normalize(point_world_pos - camera_world_pos);
    ray_origin = point_world_pos - truncation_distance * ray_direction;
  } else // (mode == RAY_DIRECTION_NORMAL)
  {
    ray_origin = point_world_pos - truncation_distance * normal_world;
    ray_direction = normal_world;
  }

  float normalized_depth = geometry_helper.NormalizeDepth(
      depth,
      sensor_params.min_depth_range,
      sensor_params.max_depth_range
  );
  float directional_weights[N_DIRECTIONS];
  ComputeDirectionWeights(normal_world, directional_weights);

  BlockTraversal voxel_traversal(
      ray_origin,
      ray_direction,
      2 * truncation_distance, // 2 * truncation, because it covers both positive and negative range
      geometry_helper.voxel_size);
  while (voxel_traversal.HasNextBlock())
  {
    const size_t voxel_array_idx = 0;

    int3 voxel_idx = voxel_traversal.GetNextBlock();
    if (runtime_params.enable_directional_sdf)
    {
      for (size_t direction = 0; direction < N_DIRECTIONS; direction++)
      {
        if (directional_weights[direction] > direction_weight_threshold)
        {

          UpdateVoxel(
              voxel_idx,
              direction,
              normalized_depth,
              truncation_distance,
              point_world_pos,
              normal_world,
              normal_camera,
              blocks,
              sensor_params,
              hash_table,
              geometry_helper,
              runtime_params.enable_point_to_plane
          );
        }
      }
    } else
    {
      UpdateVoxel(
          voxel_idx,
          voxel_array_idx,
          normalized_depth,
          truncation_distance,
          point_world_pos,
          normal_world,
          normal_camera,
          blocks,
          sensor_params,
          hash_table,
          geometry_helper,
          runtime_params.enable_point_to_plane
      );
    }

  }
}

/**
 * For every updated Voxel compute perform the SDF update (take summation values, perform single update)
 * @param candidate_entries
 * @param num_entries
 * @param blocks
 */
__global__
void UpdateRaycastedBlocksKernel(
    EntryArray candidate_entries,
    uint num_entries,
    BlockArray blocks,
    RuntimeParams runtime_params)
{
  size_t idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (idx >= num_entries)
  {
    return;
  }
  const HashEntry &entry = candidate_entries[idx];

  size_t max_voxel_idx = 0;
  if (runtime_params.enable_directional_sdf)
    max_voxel_idx = 5;
  for (size_t direction = 0; direction <= max_voxel_idx; direction++)
  {
    if (not blocks.HasVoxelArray(entry.ptr, direction))
    {
      continue;
    }
    VoxelArray &voxel_array = blocks.GetVoxelArray(entry.ptr, direction);
    for (size_t i = 0; i < BLOCK_SIZE; i++)
    {
      Voxel &voxel = voxel_array.voxels[i];
      if (voxel.num_updates == 0)
        continue;
      Voxel delta;
      delta.sdf = voxel.a / voxel.b;
      delta.inv_sigma2 = voxel.b; // / voxel.num_updates;

      voxel_array.voxels[i].Update(delta);

      // Reset summation values for next iteration
      voxel_array.voxels[i].a = 0;
      voxel_array.voxels[i].b = 0;
      voxel_array.voxels[i].num_updates = 0;
    }
  }
}

struct VoxelUpdateStatistics
{
  /** Mean number of updates per voxel (all voxels) */
  float num_updates_mean;
  /** Mean number of updates per voxels that are actually updated */
  float num_updates_hit_mean;
  /** Max number of updates per voxels that are actually updated */
  int num_updates_hit_min;
  /** Min number of updates per voxels that are actually updated */
  int num_updates_hit_max;
};

__global__
void CollectUpdateStatisticsKernel(
    EntryArray candidate_entries,
    uint num_entries,
    BlockArray blocks,
    RuntimeParams runtime_params,
    VoxelUpdateStatistics *stats
)
{
  size_t idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (idx >= num_entries)
  {
    return;
  }
  stats[idx].num_updates_hit_max = -1;
  stats[idx].num_updates_hit_min = -1;
  stats[idx].num_updates_hit_mean = 0;
  stats[idx].num_updates_mean = 0;

  const HashEntry &entry = candidate_entries[idx];
  size_t max_voxel_idx = 0;
  if (runtime_params.enable_directional_sdf)
    max_voxel_idx = 5;
  int count = 0;
  int count_hit = 0;
  for (size_t direction = 0; direction <= max_voxel_idx; direction++)
  {
    if (not blocks.HasVoxelArray(entry.ptr, direction))
    {
      continue;
    }
    VoxelArray &voxel_array = blocks.GetVoxelArray(entry.ptr, direction);
    for (size_t i = 0; i < BLOCK_SIZE; i++)
    {
      Voxel &voxel = voxel_array.voxels[i];
      stats[idx].num_updates_mean += voxel.num_updates;
      if (voxel.num_updates)
      {
        stats[idx].num_updates_hit_mean += voxel.num_updates;
        stats[idx].num_updates_hit_max = max(stats[idx].num_updates_hit_max, (int) voxel.num_updates);
        if (stats[idx].num_updates_hit_min == -1)
          stats[idx].num_updates_hit_min = voxel.num_updates;
        else
          stats[idx].num_updates_hit_min = min(stats[idx].num_updates_hit_min, voxel.num_updates);
        count_hit++;
      }
      count++;
    }
  }
  stats[idx].num_updates_mean /= count;
  stats[idx].num_updates_hit_mean /= count_hit;
}

void CollectUpdateStatistics(
    EntryArray candidate_entries,
    uint num_entries,
    BlockArray blocks,
    RuntimeParams runtime_params)
{
  static bool initialized = false;
  VoxelUpdateStatistics *stats, *stats_cpu;

  checkCudaErrors(hipMalloc(&stats, sizeof(VoxelUpdateStatistics) * num_entries));
  stats_cpu = (VoxelUpdateStatistics *) malloc(sizeof(VoxelUpdateStatistics) * num_entries);

  const dim3 num_blocks_alloc(static_cast<unsigned int>(
                                  std::ceil(num_entries / static_cast<double>(CUDA_THREADS_PER_BLOCK))));
  const dim3 num_threads_alloc(CUDA_THREADS_PER_BLOCK);
  CollectUpdateStatisticsKernel << < num_blocks_alloc, num_threads_alloc >> > (
      candidate_entries,
          num_entries,
          blocks,
          runtime_params,
          stats);
  checkCudaErrors(
      hipMemcpy(stats_cpu, stats, num_entries * sizeof(VoxelUpdateStatistics), hipMemcpyDeviceToHost));
  int total_hit_max = 0;
  int total_hit_min = std::numeric_limits<int>::max();
  float total_hit_mean = 0;
  float total_mean = 0;
  for (size_t i = 0; i < num_entries; i++)
  {
    total_hit_max = max(total_hit_max, stats_cpu[i].num_updates_hit_max);
    if (stats_cpu[i].num_updates_hit_min > 0)
      total_hit_min = min(total_hit_min, stats_cpu[i].num_updates_hit_min);
    total_mean += stats_cpu[i].num_updates_mean / num_entries;
    total_hit_mean += stats_cpu[i].num_updates_hit_mean / num_entries;
  }
  free(stats_cpu);
  checkCudaErrors(hipFree(stats));
  printf("MIN: %i, MAX: %i, MEAN(hit): %f, MEAN(total): %f\n", total_hit_min, total_hit_max, total_hit_mean,
         total_mean);
}

double UpdateRaycasting(
    EntryArray &candidate_entries,
    BlockArray &blocks,
    Sensor &sensor,
    const RuntimeParams &runtime_params,
    HashTable &hash_table,
    GeometryHelper &geometry_helper
)
{
  Timer timer;
  timer.Tick();

  uint candidate_entry_count = candidate_entries.count();
  if (candidate_entry_count <= 0)
    return timer.Tock();

  /// 1) Fuse depth data
  const int threads_per_direction = 16;
  const dim3 grid_size_fusion((sensor.width() + threads_per_direction - 1) / threads_per_direction,
                              (sensor.height() + threads_per_direction - 1) / threads_per_direction);
  const dim3 block_size_fusion(threads_per_direction, threads_per_direction);
  UpdateRaycastingKernel << < grid_size_fusion, block_size_fusion >> > (
      blocks,
          sensor.data(),
          sensor.sensor_params(),
          runtime_params,
          sensor.wTc(),
          hash_table,
          geometry_helper);
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipGetLastError());

  CollectUpdateStatistics(candidate_entries, candidate_entry_count, blocks, runtime_params);

  /// 2) Update SDF with fused values
  const dim3 num_blocks_alloc(static_cast<unsigned int>(
                                  std::ceil(candidate_entry_count / static_cast<double>(CUDA_THREADS_PER_BLOCK))));
  const dim3 num_threads_alloc(CUDA_THREADS_PER_BLOCK);
  UpdateRaycastedBlocksKernel << < num_blocks_alloc, num_threads_alloc >> > (
      candidate_entries,
          candidate_entry_count,
          blocks,
          runtime_params
  );
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipGetLastError());

  return timer.Tock();
}
