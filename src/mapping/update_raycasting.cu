#include "hip/hip_runtime.h"
#include "core/directional_tsdf.h"
#include "core/functions.h"
#include "engine/main_engine.h"
#include "mapping/allocate.h"
#include "mapping/block_traversal.hpp"
#include "mapping/update_raycasting.h"
#include "mapping/weight_functions.h"
#include "util/timer.h"
#include "geometry/geometry_helper.h"

////////////////////
/// Device code
////////////////////

__device__
inline void CarveVoxel(
    const int3 &voxel_idx,
    const size_t voxel_array_idx,
    const float normalized_depth,
    const float truncation_distance,
    const float3 &surface_point_world,
    const float3 &normal_world,
    const float4 &normal_camera,
    const BlockArray &blocks,
    const SensorParams &sensor_params,
    const HashTable &hash_table,
    GeometryHelper &geometry_helper,
    const bool enable_point_to_plane
)
{
  int3 block_idx = geometry_helper.VoxelToBlock(voxel_idx);
  uint local_idx = geometry_helper.VectorizeOffset(geometry_helper.VoxelToOffset(block_idx, voxel_idx));
  Voxel &voxel = blocks.GetVoxelArray(hash_table.GetEntry(block_idx).ptr, voxel_array_idx).voxels[local_idx];

  if (voxel.inv_sigma2 <= 0)
    return;

  float sdf = 1;
//  float weight = 1;
  float weight = fmaxf(geometry_helper.weight_sample *
                       weight_depth(normalized_depth) *
                       weight_normal_angle(make_float3(normal_camera)), 1.0f);

  atomicAdd(&voxel.a, weight * sdf);
  atomicAdd(&voxel.b, weight);
  atomicAdd(&voxel.num_updates, 1);
  return;
}

__device__
inline void UpdateVoxel(
    const int3 &voxel_idx,
    const size_t voxel_array_idx,
    const float normalized_depth,
    const float truncation_distance,
    const float3 &surface_point_world,
    const float3 &normal_world,
    const float4 &normal_camera,
    const BlockArray &blocks,
    const SensorParams &sensor_params,
    const HashTable &hash_table,
    GeometryHelper &geometry_helper,
    const bool enable_point_to_plane
)
{
  int3 block_idx = geometry_helper.VoxelToBlock(voxel_idx);
  uint local_idx = geometry_helper.VectorizeOffset(geometry_helper.VoxelToOffset(block_idx, voxel_idx));

  if (not blocks.HasVoxelArray(hash_table.GetEntry(block_idx).ptr, voxel_array_idx))
  {
    // FIXME: Issues from parallel allocation
//    printf("(%i, %i, %i) ", voxel_idx.x, voxel_idx.y, voxel_idx.z);
    return;
  }
  Voxel &voxel = blocks.GetVoxelArray(hash_table.GetEntry(block_idx).ptr, voxel_array_idx).voxels[local_idx];

  float3 voxel_pos_world = geometry_helper.VoxelToWorld(voxel_idx);

                     DirectionWeight(DirectionAngle(normal_world, voxel_array_idx));
  float weight = fmaxf(geometry_helper.weight_sample *
                       weight_depth(normalized_depth) *
                       //                       weight_voxel_correlation(surface_point_world, voxel_pos_world, truncation_distance) *
                       weight_normal_angle(make_float3(normal_camera)) *

//                       DirectionWeight(DirectionAngle(normal_world, voxel_array_idx)),
                       weight_direction_compliance(voxel_array_idx, normal_world),
                       1.0f);

  float3 observation_ray = voxel_pos_world - surface_point_world;
  float sdf;
  if (enable_point_to_plane)
    sdf = dot(observation_ray, normal_world);
  else
    sdf = sign(dot(observation_ray, normal_world)) * length(observation_ray);

  atomicAdd(&voxel.a, weight * sdf);
  atomicAdd(&voxel.b, weight);
  atomicAdd(&voxel.num_updates, 1);
}

/**
 * For every pixel casts a ray into multiple voxels (truncation range and updates the
 * SDF and weight summation values)
 * @param blocks
 * @param sensor_data
 * @param sensor_params
 * @param runtime_params
 * @param wTc
 * @param hash_table
 * @param geometry_helper
 */
__global__
void UpdateRaycastingKernel(
    SensorData sensor_data,
    SensorParams sensor_params,
    float4x4 wTc,
    BlockArray blocks,
    RuntimeParams runtime_params,
    HashTable hash_table,
    GeometryHelper geometry_helper
)
{
  // Pixel coordinates
  const int ux = blockIdx.x * blockDim.x + threadIdx.x;
  const int uy = blockIdx.y * blockDim.y + threadIdx.y;

  if (ux >= sensor_params.width || uy >= sensor_params.height)
    return;

  float depth = tex2D<float>(sensor_data.depth_texture, ux, uy);
  float4 normal_camera = tex2D<float4>(sensor_data.normal_texture, ux, uy);

  if (not IsValidDepth(depth) or depth >= geometry_helper.sdf_upper_bound or not IsValidNormal(normal_camera))
    return;

  float3 point_camera_pos = GeometryHelper::ImageReprojectToCamera(ux, uy, depth,
                                                                   sensor_params.fx, sensor_params.fy,
                                                                   sensor_params.cx, sensor_params.cy);
  float3 point_world_pos = make_float3(wTc * make_float4(point_camera_pos, 1));

  float4x4 wTcRotOnly = wTc;
  wTcRotOnly.m14 = 0;
  wTcRotOnly.m24 = 0;
  wTcRotOnly.m34 = 0;
  float3 normal_world = make_float3(wTcRotOnly * normal_camera);

  float truncation_distance = geometry_helper.truncate_distance(depth);

  // Traverse voxels in normal's direction through measured surface point
  float3 ray_origin;
  float3 ray_direction_before;
  float3 ray_direction_behind;

  if (runtime_params.raycasting_mode == RAY_DIRECTION_CAMERA)
  {
    float3 camera_world_pos = make_float3(wTc * make_float4(0, 0, 0, 1));
    ray_direction_before = ray_direction_behind = normalize(point_world_pos - camera_world_pos);
    ray_origin = point_world_pos - truncation_distance * ray_direction_before;
  }
  if (runtime_params.raycasting_mode == RAY_DIRECTION_POS_CAMERA_NEG_NORMAL)
  {
    float3 camera_world_pos = make_float3(wTc * make_float4(0, 0, 0, 1));
    ray_direction_before = normalize(point_world_pos - camera_world_pos);
    ray_direction_behind = -normal_world;
    ray_origin = point_world_pos - truncation_distance * ray_direction_before;
  } else // (runtime_params.raycasting_mode == RAY_DIRECTION_NORMAL)
  {
    ray_direction_before = normal_world;
    ray_origin = point_world_pos - truncation_distance * ray_direction_before;
  }

  float normalized_depth = geometry_helper.NormalizeDepth(
      depth,
      sensor_params.min_depth_range,
      sensor_params.max_depth_range
  );
  float directional_weights[N_DIRECTIONS];
  ComputeDirectionWeights(normal_world, directional_weights);


  /////////////////////////////////// voxel carving
#if 0
//  float3 carving_origin = make_float3(wTc * make_float4(normalize(point_camera_pos) * 0.5, 1));
  float3 carving_origin = make_float3(wTc *
                                      make_float4(
                                          GeometryHelper::ImageReprojectToCamera(ux, uy, 0.5,
                                                                                 sensor_params.fx, sensor_params.fy,
                                                                                 sensor_params.cx, sensor_params.cy),
                                          1));
  float3 carving_ray_direction = normalize(point_world_pos - carving_origin);
//  float carving_distance =
//      length(point_world_pos - carving_origin) - 2 / (-normal_camera.z) * truncation_distance;
  float carving_distance =
      length(point_world_pos - carving_origin) - fabs(1.5 / dot(normal_world, carving_ray_direction)) * truncation_distance;
//  float carving_distance = length(point_world_pos - carving_origin) - truncation_distance;

  BlockTraversal voxel_traversal_carving(
      carving_origin,
      carving_ray_direction,
      carving_distance,
      geometry_helper.voxel_size);
  int3 last_block_idx = make_int3(1000000, 1000000, 1000000);
  int direction_block_exists[6];
  while (voxel_traversal_carving.HasNextBlock())
  {
    if (voxel_traversal_carving.distance > length(point_world_pos - carving_origin))
      printf("%f / %f (%f)\n", voxel_traversal_carving.distance, length(point_world_pos - carving_origin),
          carving_distance);
    int3 voxel_idx = voxel_traversal_carving.GetNextBlock();
    int3 block_idx = geometry_helper.VoxelToBlock(voxel_idx);
    if (not(block_idx == last_block_idx))
    {
      last_block_idx = block_idx;
      for (size_t direction = 0; direction < N_DIRECTIONS; direction++)
        direction_block_exists[direction] = -1;
    }

    if (runtime_params.enable_directional_sdf)
    {
      for (size_t direction = 0; direction < N_DIRECTIONS; direction++)
      {
        if (direction_block_exists[direction] == -1)
        {
          if (blocks.HasVoxelArray(hash_table.GetEntry(block_idx).ptr, direction))
            direction_block_exists[direction] = 1;
          else
            direction_block_exists[direction] = 0;
        }
        if (direction_block_exists[direction] == 0)
          continue;
        CarveVoxel(
            voxel_idx,
            direction,
            normalized_depth,
            truncation_distance,
            point_world_pos,
            normal_world,
            normal_camera,
            blocks,
            sensor_params,
            hash_table,
            geometry_helper,
            runtime_params.enable_point_to_plane
        );
      }
    } else
    {
      const size_t voxel_array_idx = 0;
      CarveVoxel(
          voxel_idx,
          voxel_array_idx,
          normalized_depth,
          truncation_distance,
          point_world_pos,
          normal_world,
          normal_camera,
          blocks,
          sensor_params,
          hash_table,
          geometry_helper,
          runtime_params.enable_point_to_plane
      );
    }

  }
#endif
  //////////////////////////////////

  BlockTraversal voxel_traversal_before(
      ray_origin,
      ray_direction_before,
      truncation_distance,
      geometry_helper.voxel_size);
  BlockTraversal voxel_traversal_behind(
      point_world_pos,
      ray_direction_behind,
      truncation_distance,
      geometry_helper.voxel_size);
  if (voxel_traversal_behind.HasNextBlock()) voxel_traversal_behind.GetNextBlock(); // Skip first voxel to prevent duplicate fusion
  while (voxel_traversal_before.HasNextBlock() or voxel_traversal_behind.HasNextBlock())
  {
    const size_t voxel_array_idx = 0;

    int3 voxel_idx;
    if (voxel_traversal_before.HasNextBlock())
      voxel_idx = voxel_traversal_before.GetNextBlock();
    else
      voxel_idx = voxel_traversal_behind.GetNextBlock();

    if (runtime_params.enable_directional_sdf)
    {
      for (size_t direction = 0; direction < N_DIRECTIONS; direction++)
      {
        if (directional_weights[direction] > 0)
        {
          UpdateVoxel(
              voxel_idx,
              direction,
              normalized_depth,
              truncation_distance,
              point_world_pos,
              normal_world,
              normal_camera,
              blocks,
              sensor_params,
              hash_table,
              geometry_helper,
              runtime_params.enable_point_to_plane
          );
        }
      }
    } else
    {
      UpdateVoxel(
          voxel_idx,
          voxel_array_idx,
          normalized_depth,
          truncation_distance,
          point_world_pos,
          normal_world,
          normal_camera,
          blocks,
          sensor_params,
          hash_table,
          geometry_helper,
          runtime_params.enable_point_to_plane
      );
    }

  }
}

/**
 * For every updated Voxel compute perform the SDF update (take summation values, perform single update)
 * @param candidate_entries
 * @param num_entries
 * @param blocks
 */
__global__
void UpdateRaycastedBlocksKernel(
    EntryArray candidate_entries,
    uint num_entries,
    BlockArray blocks,
    RuntimeParams runtime_params)
{
  size_t idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (idx >= num_entries)
  {
    return;
  }
  const HashEntry &entry = candidate_entries[idx];

  size_t max_voxel_idx = 0;
  if (runtime_params.enable_directional_sdf)
    max_voxel_idx = 5;
  for (size_t direction = 0; direction <= max_voxel_idx; direction++)
  {
    if (not blocks.HasVoxelArray(entry.ptr, direction))
    {
      continue;
    }
    VoxelArray &voxel_array = blocks.GetVoxelArray(entry.ptr, direction);
    for (size_t i = 0; i < BLOCK_SIZE; i++)
    {
      Voxel &voxel = voxel_array.voxels[i];
      if (voxel.num_updates == 0)
        continue;
      Voxel delta;
      delta.sdf = voxel.a / voxel.b;
      delta.inv_sigma2 = voxel.b; // / voxel.num_updates;

      voxel_array.voxels[i].Update(delta);

      // Reset summation values for next iteration
      voxel_array.voxels[i].a = 0;
      voxel_array.voxels[i].b = 0;
      voxel_array.voxels[i].num_updates = 0;
    }
  }
}

struct VoxelUpdateStatistics
{
  /** Mean number of updates per voxel (all voxels) */
  float num_updates_mean;
  /** Mean number of updates per voxels that are actually updated */
  float num_updates_hit_mean;
  /** Max number of updates per voxels that are actually updated */
  int num_updates_hit_min;
  /** Min number of updates per voxels that are actually updated */
  int num_updates_hit_max;
};

__global__
void CollectUpdateStatisticsKernel(
    EntryArray candidate_entries,
    uint num_entries,
    BlockArray blocks,
    RuntimeParams runtime_params,
    VoxelUpdateStatistics *stats
)
{
  size_t idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (idx >= num_entries)
  {
    return;
  }
  stats[idx].num_updates_hit_max = -1;
  stats[idx].num_updates_hit_min = -1;
  stats[idx].num_updates_hit_mean = 0;
  stats[idx].num_updates_mean = 0;

  const HashEntry &entry = candidate_entries[idx];
  size_t max_voxel_idx = 0;
  if (runtime_params.enable_directional_sdf)
    max_voxel_idx = 5;
  int count = 0;
  int count_hit = 0;
  for (size_t direction = 0; direction <= max_voxel_idx; direction++)
  {
    if (not blocks.HasVoxelArray(entry.ptr, direction))
    {
      continue;
    }
    VoxelArray &voxel_array = blocks.GetVoxelArray(entry.ptr, direction);
    for (size_t i = 0; i < BLOCK_SIZE; i++)
    {
      Voxel &voxel = voxel_array.voxels[i];
      stats[idx].num_updates_mean += voxel.num_updates;
      if (voxel.num_updates)
      {
        stats[idx].num_updates_hit_mean += voxel.num_updates;
        stats[idx].num_updates_hit_max = max(stats[idx].num_updates_hit_max, (int) voxel.num_updates);
        if (stats[idx].num_updates_hit_min == -1)
          stats[idx].num_updates_hit_min = voxel.num_updates;
        else
          stats[idx].num_updates_hit_min = min(stats[idx].num_updates_hit_min, voxel.num_updates);
        count_hit++;
      }
      count++;
    }
  }
  stats[idx].num_updates_mean /= count;
  stats[idx].num_updates_hit_mean /= count_hit;
}

void CollectUpdateStatistics(
    EntryArray candidate_entries,
    MainEngine &main_engine)
{
  uint num_entries = candidate_entries.count();

  VoxelUpdateStatistics *stats, *stats_cpu;
  checkCudaErrors(hipMalloc(&stats, sizeof(VoxelUpdateStatistics) * num_entries));
  stats_cpu = (VoxelUpdateStatistics *) malloc(sizeof(VoxelUpdateStatistics) * num_entries);

  const dim3 num_blocks_alloc(static_cast<unsigned int>(
                                  std::ceil(num_entries / static_cast<double>(CUDA_THREADS_PER_BLOCK))));
  const dim3 num_threads_alloc(CUDA_THREADS_PER_BLOCK);
  CollectUpdateStatisticsKernel << < num_blocks_alloc, num_threads_alloc >> > (
      candidate_entries,
          num_entries,
          main_engine.blocks(),
          main_engine.runtime_params(),
          stats);
  checkCudaErrors(
      hipMemcpy(stats_cpu, stats, num_entries * sizeof(VoxelUpdateStatistics), hipMemcpyDeviceToHost));
  int total_hit_max = 0;
  int total_hit_min = std::numeric_limits<int>::max();
  float total_hit_mean = 0;
  float total_mean = 0;
  for (size_t i = 0; i < num_entries; i++)
  {
    total_hit_max = max(total_hit_max, stats_cpu[i].num_updates_hit_max);
    if (stats_cpu[i].num_updates_hit_min > 0)
      total_hit_min = min(total_hit_min, stats_cpu[i].num_updates_hit_min);
    total_mean += stats_cpu[i].num_updates_mean / num_entries;
    total_hit_mean += stats_cpu[i].num_updates_hit_mean / num_entries;
  }
  free(stats_cpu);
  checkCudaErrors(hipFree(stats));
//  printf("MIN: %i, MAX: %i, MEAN(hit): %f, MEAN(total): %f\n", total_hit_min, total_hit_max, total_hit_mean,
//         total_mean);
  main_engine.log_engine().WriteVoxelUpdate(total_hit_max, total_hit_mean, total_mean);
}

double UpdateRaycasting(
    EntryArray &candidate_entries,
    Sensor &sensor,
    MainEngine &main_engine
)
{
  Timer timer;
  timer.Tick();

  uint candidate_entry_count = candidate_entries.count();
  if (candidate_entry_count <= 0)
    return timer.Tock();

  /// 1) Fuse depth data
  const int threads_per_direction = 16;
  const dim3 grid_size_fusion((sensor.width() + threads_per_direction - 1) / threads_per_direction,
                              (sensor.height() + threads_per_direction - 1) / threads_per_direction);
  const dim3 block_size_fusion(threads_per_direction, threads_per_direction);
  UpdateRaycastingKernel << < grid_size_fusion, block_size_fusion >> > (
      sensor.data(),
          sensor.sensor_params(),
          sensor.wTc(),
          main_engine.blocks(),
          main_engine.runtime_params(),
          main_engine.hash_table(),
          main_engine.geometry_helper());
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipGetLastError());

//  CollectUpdateStatistics(candidate_entries, main_engine);

  /// 2) Update SDF with fused values
  const dim3 num_blocks_alloc(static_cast<unsigned int>(
                                  std::ceil(candidate_entry_count / static_cast<double>(CUDA_THREADS_PER_BLOCK))));
  const dim3 num_threads_alloc(CUDA_THREADS_PER_BLOCK);
  UpdateRaycastedBlocksKernel << < num_blocks_alloc, num_threads_alloc >> > (
      candidate_entries,
          candidate_entry_count,
          main_engine.blocks(),
          main_engine.runtime_params()
  );
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipGetLastError());

  return timer.Tock();
}
