#include "hip/hip_runtime.h"
//
// Created by wei on 17-10-21.
//

#include "bounding_box.h"
#include "hip/hip_runtime_api.h"

////////////////////
/// class BBox
////////////////////
BBox::BBox() {}
BBox::~BBox() {
  Free();
}

void BBox::Alloc(int max_vertex_count) {
  checkCudaErrors(hipMalloc(&gpu_memory_.vertex_counter,
                             sizeof(uint)));
  checkCudaErrors(hipMalloc(&gpu_memory_.vertices,
                             sizeof(float3) * max_vertex_count));
}

void BBox::Free() {
  checkCudaErrors(hipFree(gpu_memory_.vertex_counter));
  checkCudaErrors(hipFree(gpu_memory_.vertices));
}

void BBox::Resize(int max_vertex_count) {
  max_vertex_count_ = max_vertex_count;
  Alloc(max_vertex_count);
  Reset();
}

void BBox::Reset() {
  checkCudaErrors(hipMemset(gpu_memory_.vertex_counter,
                             0, sizeof(uint)));
}

uint BBox::vertex_count() {
  uint vertex_count;
  checkCudaErrors(hipMemcpy(&vertex_count,
                             gpu_memory_.vertex_counter,
                             sizeof(uint), hipMemcpyDeviceToHost));
  return vertex_count;
}