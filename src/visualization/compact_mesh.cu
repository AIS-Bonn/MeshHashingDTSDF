#include "hip/hip_runtime.h"
//
// Created by wei on 17-10-21.
//

#include "compact_mesh.h"
#include "hip/hip_runtime_api.h"
////////////////////
/// class CompactMesh
////////////////////

/// Life cycle
CompactMesh::CompactMesh() {}

//CompactMesh::~CompactMesh() {
//  Free();
//}

void CompactMesh::Alloc(const MeshParams &mesh_params) {
  checkCudaErrors(hipMalloc(&vertex_remapper_,
                             sizeof(int) * mesh_params.max_vertex_count));

  checkCudaErrors(hipMalloc(&vertex_counter_,
                             sizeof(uint)));
  checkCudaErrors(hipMalloc(&vertices_ref_count_,
                             sizeof(int) * mesh_params.max_vertex_count));
  checkCudaErrors(hipMalloc(&vertices_,
                             sizeof(float3) * mesh_params.max_vertex_count));
  checkCudaErrors(hipMalloc(&normals_,
                             sizeof(float3) * mesh_params.max_vertex_count));
  checkCudaErrors(hipMalloc(&colors_,
                             sizeof(float3) * mesh_params.max_vertex_count));

  checkCudaErrors(hipMalloc(&triangle_counter_,
                             sizeof(uint)));
  checkCudaErrors(hipMalloc(&triangles_ref_count_,
                             sizeof(int) * mesh_params.max_triangle_count));
  checkCudaErrors(hipMalloc(&triangles_,
                             sizeof(int3) * mesh_params.max_triangle_count));
}

void CompactMesh::Free() {
  checkCudaErrors(hipFree(vertex_remapper_));

  checkCudaErrors(hipFree(vertex_counter_));
  checkCudaErrors(hipFree(vertices_ref_count_));
  checkCudaErrors(hipFree(vertices_));
  checkCudaErrors(hipFree(normals_));
  checkCudaErrors(hipFree(colors_));

  checkCudaErrors(hipFree(triangle_counter_));
  checkCudaErrors(hipFree(triangles_ref_count_));
  checkCudaErrors(hipFree(triangles_));
}

void CompactMesh::Resize(const MeshParams &mesh_params) {
  mesh_params_ = mesh_params;
  Alloc(mesh_params);
  Reset();
}

/// Reset
void CompactMesh::Reset() {
  checkCudaErrors(hipMemset(vertex_remapper_, 0xff,
                             sizeof(int) * mesh_params_.max_vertex_count));
  checkCudaErrors(hipMemset(vertices_ref_count_, 0,
                             sizeof(int) * mesh_params_.max_vertex_count));
  checkCudaErrors(hipMemset(vertex_counter_,
                             0, sizeof(uint)));
  checkCudaErrors(hipMemset(triangles_ref_count_, 0,
                             sizeof(int) * mesh_params_.max_triangle_count));
  checkCudaErrors(hipMemset(triangle_counter_,
                             0, sizeof(uint)));
}

uint CompactMesh::vertex_count() {
  uint compact_vertex_count;
  checkCudaErrors(hipMemcpy(&compact_vertex_count,
                             vertex_counter_,
                             sizeof(uint), hipMemcpyDeviceToHost));
  return compact_vertex_count;
}

uint CompactMesh::triangle_count() {
  uint compact_triangle_count;
  checkCudaErrors(hipMemcpy(&compact_triangle_count,
                             triangle_counter_,
                             sizeof(uint), hipMemcpyDeviceToHost));
  return compact_triangle_count;
}
