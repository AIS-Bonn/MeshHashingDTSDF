#include "hip/hip_runtime.h"
#include "mesh.h"

#include <hip/hip_runtime_api.h>
#include <>
#include "params.h"
#include <glog/logging.h>

////////////////////
/// class Mesh
////////////////////

////////////////////
/// Device code
////////////////////
__global__
void MeshResetVerticesKernel(uint* vertex_heap, Vertex* vertices, int max_vertex_count) {
  const uint idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < max_vertex_count) {
    vertex_heap[idx] = max_vertex_count - idx - 1;
    vertices[idx].Clear();
  }
}

__global__
void MeshResetTrianglesKernel(uint* triangle_heap, Triangle* triangles, int max_triangle_count) {
  const uint idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < max_triangle_count) {
    triangle_heap[idx] = max_triangle_count - idx - 1;
    triangles[idx].Clear();
  }
}

////////////////////
/// Host code
////////////////////
__host__ Mesh::Mesh() {}

// Mesh::~Mesh() {
  //Free();
//}

__host__
void Mesh::Alloc(const MeshParams &mesh_params) {
  checkCudaErrors(hipMalloc(&vertex_heap_,
                             sizeof(uint) * mesh_params.max_vertex_count));
  checkCudaErrors(hipMalloc(&vertex_heap_counter_, sizeof(uint)));
  checkCudaErrors(hipMalloc(&vertices,
                             sizeof(Vertex) * mesh_params.max_vertex_count));

  checkCudaErrors(hipMalloc(&triangle_heap_,
                             sizeof(uint) * mesh_params.max_triangle_count));
  checkCudaErrors(hipMalloc(&triangle_heap_counter_, sizeof(uint)));
  checkCudaErrors(hipMalloc(&triangles,
                             sizeof(Triangle) * mesh_params.max_triangle_count));
}

void Mesh::Free() {
  checkCudaErrors(hipFree(vertex_heap_));
  checkCudaErrors(hipFree(vertex_heap_counter_));
  checkCudaErrors(hipFree(vertices));

  checkCudaErrors(hipFree(triangle_heap_));
  checkCudaErrors(hipFree(triangle_heap_counter_));
  checkCudaErrors(hipFree(triangles));
}

void Mesh::Resize(const MeshParams &mesh_params) {
  mesh_params_ = mesh_params;
  Alloc(mesh_params);
  Reset();
}

void Mesh::Reset() {
  uint val;

  val = mesh_params_.max_vertex_count - 1;
  checkCudaErrors(hipMemcpy(vertex_heap_counter_,
                             &val,
                             sizeof(uint),
                             hipMemcpyHostToDevice));

  val = mesh_params_.max_triangle_count - 1;
  checkCudaErrors(hipMemcpy(triangle_heap_counter_,
                             &val,
                             sizeof(uint),
                             hipMemcpyHostToDevice));

  {
    const int threads_per_block = 64;
    const dim3 grid_size((mesh_params_.max_vertex_count + threads_per_block - 1)
                         / threads_per_block, 1);
    const dim3 block_size(threads_per_block, 1);

    MeshResetVerticesKernel<<< grid_size, block_size >>> (vertex_heap_, vertices,
        mesh_params_.max_vertex_count);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());
  }

  {
    const int threads_per_block = 64;
    const dim3 grid_size((mesh_params_.max_triangle_count + threads_per_block - 1)
                         / threads_per_block, 1);
    const dim3 block_size(threads_per_block, 1);

    MeshResetTrianglesKernel<<<grid_size, block_size>>> (triangle_heap_, triangles,
        mesh_params_.max_triangle_count);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());
  }
}

uint Mesh::vertex_heap_count() {
  uint vertex_heap_count;
  checkCudaErrors(hipMemcpy(&vertex_heap_count,
                             vertex_heap_counter_,
                             sizeof(uint), hipMemcpyDeviceToHost));
  return vertex_heap_count;
}

uint Mesh::triangle_heap_count() {
  uint triangle_heap_count;
  checkCudaErrors(hipMemcpy(&triangle_heap_count,
                             triangle_heap_counter_,
                             sizeof(uint), hipMemcpyDeviceToHost));
  return triangle_heap_count;
}

////////////////////
/// class CompactMesh
////////////////////

/// Life cycle
CompactMesh::CompactMesh() {}

CompactMesh::~CompactMesh() {
  Free();
}

void CompactMesh::Alloc(const MeshParams &mesh_params) {
  checkCudaErrors(hipMalloc(&gpu_memory_.vertex_remapper,
                             sizeof(int) * mesh_params.max_vertex_count));

  checkCudaErrors(hipMalloc(&gpu_memory_.vertex_counter,
                             sizeof(uint)));
  checkCudaErrors(hipMalloc(&gpu_memory_.vertices_ref_count,
                             sizeof(int) * mesh_params.max_vertex_count));
  checkCudaErrors(hipMalloc(&gpu_memory_.vertices,
                             sizeof(float3) * mesh_params.max_vertex_count));
  checkCudaErrors(hipMalloc(&gpu_memory_.normals,
                             sizeof(float3) * mesh_params.max_vertex_count));
  checkCudaErrors(hipMalloc(&gpu_memory_.colors,
                             sizeof(float3) * mesh_params.max_vertex_count));

  checkCudaErrors(hipMalloc(&gpu_memory_.triangle_counter,
                             sizeof(uint)));
  checkCudaErrors(hipMalloc(&gpu_memory_.triangles_ref_count,
                             sizeof(int) * mesh_params.max_triangle_count));
  checkCudaErrors(hipMalloc(&gpu_memory_.triangles,
                             sizeof(int3) * mesh_params.max_triangle_count));
}

void CompactMesh::Free() {
  checkCudaErrors(hipFree(gpu_memory_.vertex_remapper));

  checkCudaErrors(hipFree(gpu_memory_.vertex_counter));
  checkCudaErrors(hipFree(gpu_memory_.vertices_ref_count));
  checkCudaErrors(hipFree(gpu_memory_.vertices));
  checkCudaErrors(hipFree(gpu_memory_.normals));
  checkCudaErrors(hipFree(gpu_memory_.colors));

  checkCudaErrors(hipFree(gpu_memory_.triangle_counter));
  checkCudaErrors(hipFree(gpu_memory_.triangles_ref_count));
  checkCudaErrors(hipFree(gpu_memory_.triangles));
}

void CompactMesh::Resize(const MeshParams &mesh_params) {
  mesh_params_ = mesh_params;
  Alloc(mesh_params);
  Reset();
}

/// Reset
void CompactMesh::Reset() {
  checkCudaErrors(hipMemset(gpu_memory_.vertex_remapper, 0xff,
                             sizeof(int) * mesh_params_.max_vertex_count));
  checkCudaErrors(hipMemset(gpu_memory_.vertices_ref_count, 0,
                             sizeof(int) * mesh_params_.max_vertex_count));
  checkCudaErrors(hipMemset(gpu_memory_.vertex_counter,
                             0, sizeof(uint)));
  checkCudaErrors(hipMemset(gpu_memory_.triangles_ref_count, 0,
                             sizeof(int) * mesh_params_.max_triangle_count));
  checkCudaErrors(hipMemset(gpu_memory_.triangle_counter,
                             0, sizeof(uint)));
}

uint CompactMesh::vertex_count() {
  uint compact_vertex_count;
  checkCudaErrors(hipMemcpy(&compact_vertex_count,
                             gpu_memory_.vertex_counter,
                             sizeof(uint), hipMemcpyDeviceToHost));
  return compact_vertex_count;
}

uint CompactMesh::triangle_count() {
  uint compact_triangle_count;
  checkCudaErrors(hipMemcpy(&compact_triangle_count,
                             gpu_memory_.triangle_counter,
                             sizeof(uint), hipMemcpyDeviceToHost));
  return compact_triangle_count;
}

////////////////////
/// class BBox
////////////////////
BBox::BBox() {}
BBox::~BBox() {
  Free();
}

void BBox::Alloc(int max_vertex_count) {
  checkCudaErrors(hipMalloc(&gpu_memory_.vertex_counter,
                             sizeof(uint)));
  checkCudaErrors(hipMalloc(&gpu_memory_.vertices,
                             sizeof(float3) * max_vertex_count));
}

void BBox::Free() {
  checkCudaErrors(hipFree(gpu_memory_.vertex_counter));
  checkCudaErrors(hipFree(gpu_memory_.vertices));
}

void BBox::Resize(int max_vertex_count) {
  max_vertex_count_ = max_vertex_count;
  Alloc(max_vertex_count);
  Reset();
}

void BBox::Reset() {
  checkCudaErrors(hipMemset(gpu_memory_.vertex_counter,
                             0, sizeof(uint)));
}

uint BBox::vertex_count() {
  uint vertex_count;
  checkCudaErrors(hipMemcpy(&vertex_count,
                             gpu_memory_.vertex_counter,
                             sizeof(uint), hipMemcpyDeviceToHost));
  return vertex_count;
}