#include "hip/hip_runtime.h"
#include "core/directional_tsdf.h"
#include "core/functions.h"
#include "meshing/mc_tables.h"

__device__
short FilterMCIndexDirection(const short mc_index, const TSDFDirection direction, const float sdf[8])
{
  if (mc_index <= 0 or mc_index == 255)
    return mc_index;

  short new_index = 0;
  for (int component = 0; component < 4 and kIndexDecomposition[mc_index][component] != -1; component++)
  {
    const short part_idx = kIndexDecomposition[mc_index][component];
    if (not IsMCIndexDirectionCompatible(part_idx, direction, sdf))
      continue;
    new_index |= part_idx;
  }

  if (new_index == 0)
  { // If 0 after filtering -> invalidate, so it doesn't affect other directions during later filtering process
    new_index = -1;
  }
  return new_index;
}


__device__
bool IsMCIndexDirectionCompatible(const short mc_index, const TSDFDirection direction, const float sdf[8])
{
  // Table containing for each direction:
  // 4 opposite edge pairs, each of which is checked individually.
  const static size_t view_direction_edges_to_check[6][8] = {
      {0, 4, 1, 5, 2,  6,  3,  7},  // Y_POS
      {4, 0, 5, 1, 6,  2,  7,  3},  // Y_NEG
      {1, 3, 5, 7, 9,  8,  10, 11}, // X_POS
      {3, 1, 7, 5, 8,  9,  11, 10}, // X_NEG
      {2, 0, 6, 4, 10, 9,  11, 8},  // Z_NEG
      {0, 2, 4, 6, 8,  11, 9,  10}  // Z_POS
  };
  if (kIndexDirectionCompatibility[mc_index][static_cast<size_t>(direction)] == 0)
    return false;
  if (kIndexDirectionCompatibility[mc_index][static_cast<size_t>(direction)] == 2)
  {
    for (int e = 0; e < 4; e++)
    {
      const size_t edge_idx = view_direction_edges_to_check[static_cast<size_t>(direction)][2 * e];
      const size_t opposite_edge_idx = view_direction_edges_to_check[static_cast<size_t>(direction)][2 * e + 1];
      int2 edge = kEdgeEndpointVertices[edge_idx];
      int2 opposite_edge = kEdgeEndpointVertices[opposite_edge_idx];

      int2 endpoint_values;
      endpoint_values.x = (mc_index & (1 << edge.x)) > 0;
      endpoint_values.y = (mc_index & (1 << edge.y)) > 0;

      // If edge has NO zero-crossing -> continue
      if (endpoint_values.x + endpoint_values.y != 1)
        continue;

      // Swap vertex indices, s.t. first endpoint is behind the surface
      if (endpoint_values.y == 1)
      {
        int tmp;
        tmp = edge.x;
        edge.x = edge.y;
        edge.y = tmp;
        tmp = opposite_edge.x;
        opposite_edge.x = opposite_edge.y;
        opposite_edge.y = tmp;
      }

      float offset = InterpolateSurfaceOffset(sdf[edge.x], sdf[edge.y], 0);
      float opposite_offset = InterpolateSurfaceOffset(sdf[opposite_edge.x], sdf[opposite_edge.y], 0);

      // If interpolated surface more than 90 degrees from view direction vector -> discard
      if (offset > opposite_offset)
      {
        return false;
      }
//      if (fabs(opposite_offset - offset) < 0.5)
//      {
//        return false;
//      }
    }
  }
  return true;
}

const char *TSDFDirectionToString(TSDFDirection direction)
{
  switch (direction)
  {
    case TSDFDirection::UP:
      return "UP";
    case TSDFDirection::DOWN:
      return "DOWN";
    case TSDFDirection::RIGHT:
      return "RIGHT";
    case TSDFDirection::FORWARD:
      return "FORWARD";
    case TSDFDirection::LEFT:
      return "LEFT";
    case TSDFDirection::BACKWARD:
      return "BACKWARD";
    default:
      return "ERROR/UNKNOWN";
  }
}

__device__
float DirectionAngle(const float3& normal, size_t direction)
{
  float angleCos = dot(normal, TSDFDirectionVectors[direction]);
  angleCos = fmaxf(fminf(angleCos, 1), -1);
  return acos(angleCos);
}

__device__
float DirectionWeight(float angle)
{
  float width = direction_angle_threshold;

  if (width <= M_PI_4 + 1e-6)
  {
    return 1 - fminf(angle / width, 1);
  }

  width /= M_PI_2;
  angle /= M_PI_2;
  return 1 - fminf((fmaxf(angle, 1 - width) - (1 - width)) / (2 * width - 1), 1);
}

__device__
void ComputeDirectionWeights(const float3 &normal, float *weights)
{
  for (size_t i = 0; i < 3; i++)
  {
    float angle = DirectionAngle(normal, 2 * i);
    weights[2 * i] = DirectionWeight(angle);
    weights[2 * i + 1] = DirectionWeight(M_PI - angle); // opposite direction -> negative value
  }
}
