#include "hip/hip_runtime.h"
#include "matrix.h"

#include "engine/main_engine.h"
#include "sensor/rgbd_sensor.h"

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

#include <unordered_set>
#include <vector>
#include <list>
#include <glog/logging.h>
#include <>
#include "meshing/mc_tables.h"


#define PINF  __int_as_float(0x7f800000)

////////////////////
/// class MappingEngine - compress, recycle
////////////////////

/// Condition: IsBlockInCameraFrustum
__global__
void CollectBlocksInFrustumKernel(HashTable hash_table,
                                  EntryArray candidate_entries,
                                  SensorParams sensor_params,
                                  float4x4 c_T_w,
                                  CoordinateConverter converter) {
  const uint idx = blockIdx.x * blockDim.x + threadIdx.x;

  __shared__ int local_counter;
  if (threadIdx.x == 0) local_counter = 0;
  __syncthreads();

  int addr_local = -1;
  if (idx < hash_table.entry_count
    && hash_table.entry(idx).ptr != FREE_ENTRY
    && converter.IsBlockInCameraFrustum(c_T_w, hash_table.entry(idx).pos,
                                        sensor_params)) {
    addr_local = atomicAdd(&local_counter, 1);
  }
  __syncthreads();

  __shared__ int addr_global;
  if (threadIdx.x == 0 && local_counter > 0) {
    addr_global = atomicAdd(&candidate_entries.counter(),
                            local_counter);
  }
  __syncthreads();

  if (addr_local != -1) {
    const uint addr = addr_global + addr_local;
    candidate_entries[addr] = hash_table.entry(idx);
  }
}

__global__
void CollectAllBlocksKernel(HashTable hash_table,
                            EntryArray candidate_entries) {
  const uint idx = blockIdx.x * blockDim.x + threadIdx.x;

  __shared__ int local_counter;
  if (threadIdx.x == 0) local_counter = 0;
  __syncthreads();

  int addr_local = -1;
  if (idx < hash_table.entry_count
      && hash_table.entry(idx).ptr != FREE_ENTRY) {
    addr_local = atomicAdd(&local_counter, 1);
  }

  __syncthreads();

  __shared__ int addr_global;
  if (threadIdx.x == 0 && local_counter > 0) {
    addr_global = atomicAdd(&candidate_entries.counter(),
                            local_counter);
  }
  __syncthreads();

  if (addr_local != -1) {
    const uint addr = addr_global + addr_local;
    candidate_entries[addr] = hash_table.entry(idx);
  }
}

////////////////////
/// Host code
///////////////////

/// Compress discrete hash table entries
void CollectAllBlocks(EntryArray &candidate_entries, HashTable &hash_table) {
  const uint threads_per_block = 256;

  uint entry_count = hash_table.entry_count;
  const dim3 grid_size((entry_count + threads_per_block - 1)
                       / threads_per_block, 1);
  const dim3 block_size(threads_per_block, 1);

  candidate_entries.reset_count();
  CollectAllBlocksKernel <<<grid_size, block_size >>>(
          hash_table,
          candidate_entries);
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipGetLastError());

  LOG(INFO) << "Block count in all: "
            << candidate_entries.count();
}

void CollectBlocksInFrustum(HashTable &hash_table,
                            EntryArray &candidate_entries,
                            Sensor &sensor,
                            CoordinateConverter &converter) {
  const uint threads_per_block = 256;

  uint entry_count = hash_table.entry_count;

  const dim3 grid_size((entry_count + threads_per_block - 1)
                       / threads_per_block, 1);
  const dim3 block_size(threads_per_block, 1);

  candidate_entries.reset_count();
  CollectBlocksInFrustumKernel <<<grid_size, block_size >>>(
      hash_table,
          candidate_entries,
          sensor.sensor_params(),
          sensor.c_T_w(),
          converter);

  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipGetLastError());

  LOG(INFO) << "Block count in view frustum: "
            << candidate_entries.count();
}

