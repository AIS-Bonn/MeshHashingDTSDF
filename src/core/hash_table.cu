#include "hip/hip_runtime.h"
#include <unordered_set>
#include <vector>
#include <list>
#include <glog/logging.h>
#include <>

#include "core/hash_table.h"

////////////////////
/// class HashTable
////////////////////

////////////////////
/// Device code
////////////////////
__global__
void ResetBucketMutexesKernel(int* bucket_mutexes, uint bucket_count) {
  const uint idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < bucket_count) {
    bucket_mutexes[idx] = FREE_ENTRY;
  }
}

__global__
void ResetHeapKernel(uint* heap, uint value_capacity) {
  const uint idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < value_capacity) {
    heap[idx] = value_capacity - idx - 1;
  }
}

__global__
void ResetEntriesKernel(HashEntry* entries, uint entry_count) {
  const uint idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < entry_count) {
    entries[idx].Clear();
  }
}

////////////////////
/// Host code
////////////////////

/// Life cycle
HashTable::HashTable() {}

HashTable::HashTable(const HashParams &params) {
  hash_params_ = params;
  Alloc(params);
  Reset();
}

//HashTable::~HashTable() {
//  Free();
//}

void HashTable::Alloc(const HashParams &params) {
  /// Parameters
  bucket_count = params.bucket_count;
  bucket_size = params.bucket_size;
  entry_count = params.entry_count;
  value_capacity = params.value_capacity;
  linked_list_size = params.linked_list_size;

  /// Values
  checkCudaErrors(hipMalloc(&heap,
                             sizeof(uint) * params.value_capacity));
  checkCudaErrors(hipMalloc(&heap_counter,
                             sizeof(uint)));

  /// Entries
  checkCudaErrors(hipMalloc(&entries,
                             sizeof(HashEntry) * params.entry_count));

  /// Mutexes
  checkCudaErrors(hipMalloc(&bucket_mutexes,
                             sizeof(int) * params.bucket_count));
}

void HashTable::Free() {
  checkCudaErrors(hipFree(heap));
  checkCudaErrors(hipFree(heap_counter));

  checkCudaErrors(hipFree(entries));
  checkCudaErrors(hipFree(bucket_mutexes));
}

void HashTable::Resize(const HashParams &params) {
  hash_params_ = params;
  Alloc(params);
  Reset();
}
/// Reset
void HashTable::Reset() {
  /// Reset mutexes
  ResetMutexes();

  {
    /// Reset entries
    const int threads_per_block = 64;
    const dim3 grid_size((hash_params_.entry_count + threads_per_block - 1)
                         / threads_per_block, 1);
    const dim3 block_size(threads_per_block, 1);

    ResetEntriesKernel<<<grid_size, block_size>>>(entries, entry_count);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());
  }

  {
    /// Reset allocated memory
    uint heap_counter_init = hash_params_.value_capacity - 1;
    checkCudaErrors(hipMemcpy(heap_counter, &heap_counter_init,
                               sizeof(uint),
                               hipMemcpyHostToDevice));

    const int threads_per_block = 64;
    const dim3 grid_size((hash_params_.value_capacity + threads_per_block - 1)
                         / threads_per_block, 1);
    const dim3 block_size(threads_per_block, 1);

    ResetHeapKernel<<<grid_size, block_size>>>(heap, value_capacity);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());
  }
}

void HashTable::ResetMutexes() {
  const int threads_per_block = 64;
  const dim3 grid_size((hash_params_.bucket_count + threads_per_block - 1)
                       / threads_per_block, 1);
  const dim3 block_size(threads_per_block, 1);

  ResetBucketMutexesKernel<<<grid_size, block_size>>>(bucket_mutexes, bucket_count);
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipGetLastError());
}

/// Member function: Others
//void HashTable::Debug() {
//  HashEntry *entries = new HashEntry[hash_params_.bucket_size * hash_params_.bucket_count];
//  uint *heap = new uint[hash_params_.value_capacity];
//  uint  heap_counter;
//
//  checkCudaErrors(hipMemcpy(&heap_counter, heap_counter, sizeof(uint), hipMemcpyDeviceToHost));
//  heap_counter++; //points to the first free entry: number of blocks is one more
//
//  checkCudaErrors(hipMemcpy(heap, heap,
//                             sizeof(uint) * hash_params_.value_capacity,
//                             hipMemcpyDeviceToHost));
//  checkCudaErrors(hipMemcpy(entries, entries,
//                             sizeof(HashEntry) * hash_params_.bucket_size * hash_params_.bucket_count,
//                             hipMemcpyDeviceToHost));
////  checkCudaErrors(hipMemcpy(values, values,
////                             sizeof(T) * hash_params_.value_capacity,
////                             hipMemcpyDeviceToHost));
//
//  LOG(INFO) << "GPU -> CPU data transfer finished";
//
//  //Check for duplicates
//  class Entry {
//  public:
//    Entry() {}
//    Entry(int x_, int y_, int z_, int i_, int offset_, int ptr_) :
//            x(x_), y(y_), z(z_), i(i_), offset(offset_), ptr(ptr_) {}
//    ~Entry() {}
//
//    bool operator< (const Entry &other) const {
//      if (x == other.x) {
//        if (y == other.y) {
//          return z < other.z;
//        } return y < other.y;
//      } return x < other.x;
//    }
//
//    bool operator== (const Entry &other) const {
//      return x == other.x && y == other.y && z == other.z;
//    }
//
//    int x, y, z, i;
//    int offset;
//    int ptr;
//  };
//
//  /// Iterate over free heap
//  std::unordered_set<uint> free_heap_index;
//  std::vector<int> free_value_index(hash_params_.value_capacity, 0);
//  for (uint i = 0; i < heap_counter; i++) {
//    free_heap_index.insert(heap[i]);
//    free_value_index[heap[i]] = FREE_ENTRY;
//  }
//  if (free_heap_index.size() != heap_counter) {
//    LOG(ERROR) << "Heap check invalid";
//  }
//
//  uint not_free_entry_count = 0;
//  uint not_locked_entry_count = 0;
//
//  /// Iterate over entries
//  std::list<Entry> l;
//  uint entry_count = hash_params_.entry_count;
//  for (uint i = 0; i < entry_count; i++) {
//    if (entries[i].ptr != LOCK_ENTRY) {
//      not_locked_entry_count++;
//    }
//
//    if (entries[i].ptr != FREE_ENTRY) {
//      not_free_entry_count++;
//      Entry occupied_entry(entries[i].pos.x, entries[i].pos.y, entries[i].pos.z,
//                           i, entries[i].offset, entries[i].ptr);
//      l.push_back(occupied_entry);
//
//      if (free_heap_index.find(occupied_entry.ptr) != free_heap_index.end()) {
//        LOG(ERROR) << "ERROR: ptr is on free heap, but also marked as an allocated entry";
//      }
//      free_value_index[entries[i].ptr] = LOCK_ENTRY;
//    }
//  }
//
//  /// Iterate over values
//  uint free_value_count = 0;
//  uint not_free_value_count = 0;
//  for (uint i = 0; i < hash_params_.value_capacity; i++) {
//    if (free_value_index[i] == FREE_ENTRY) {
//      free_value_count++;
//    } else if (free_value_index[i] == LOCK_ENTRY) {
//      not_free_value_count++;
//    } else {
//      LOG(ERROR) << "memory leak detected: neither free nor allocated";
//      return;
//    }
//  }
//
//  if (free_value_count + not_free_value_count == hash_params_.value_capacity)
//    LOG(INFO) << "HEAP OK!";
//  else {
//    LOG(ERROR) << "HEAP CORRUPTED";
//    return;
//  }
//
//  l.sort();
//  size_t size_before = l.size();
//  l.unique();
//  size_t size_after = l.size();
//
//
//  LOG(INFO) << "Duplicated entry count: " << size_before - size_after;
//  LOG(INFO) << "Not locked entry count: " << not_locked_entry_count;
//  LOG(INFO) << "Not free value count: " << not_free_value_count
//            << "; free value count: " << free_value_count;
//  LOG(INFO) << "not_free + free entry count: "
//            << not_free_value_count + free_value_count;
//
//  delete [] entries;
//  //delete [] values;
//  delete [] heap;
//}