#include "hip/hip_runtime.h"
#include "meshing/marching_cubes.h"
#include "geometry/gradient.h"
#include "visualization/color_util.h"
//#define REDUCTION

////////////////////
/// class MappingEngine - meshing
////////////////////

////////////////////
/// Device code
////////////////////

/// Marching Cubes
__device__
float3 VertexIntersection(const float3& p1, const float3 p2,
                          const float&  v1, const float& v2,
                          const float& isolevel) {
  if (fabs(v1 - isolevel) < 0.008) return p1;
  if (fabs(v2 - isolevel) < 0.008) return p2;
  float mu = (isolevel - v1) / (v2 - v1);

  float3 p = make_float3(p1.x + mu * (p2.x - p1.x),
                         p1.y + mu * (p2.y - p1.y),
                         p1.z + mu * (p2.z - p1.z));
  return p;
}

__device__
inline int AllocateVertexWithMutex(const HashTable &hash_table,
                                   BlockArray &blocks,
                                   Mesh& mesh,
                                   Voxel& voxel,
                                   uint& vertex_idx,
                                   const float3& vertex_pos,
                                   bool use_fine_gradient,
                                   CoordinateConverter& converter) {
  int ptr = voxel.vertex_ptrs[vertex_idx];
  if (ptr == FREE_PTR) {
    int lock = atomicExch(&voxel.vertex_mutexes[vertex_idx], LOCK_ENTRY);
    if (lock != LOCK_ENTRY) {
      ptr = mesh.AllocVertex();
    } /// Ensure that it is only allocated once
  }

  if (ptr >= 0) {
    voxel.vertex_ptrs[vertex_idx] = ptr;
    mesh.vertex(ptr).pos = vertex_pos;
    if (use_fine_gradient) {
      mesh.vertex(ptr).normal = GradientAtPoint(hash_table, blocks, vertex_pos, converter);
    }

    float sdf;
    Stat  stats;
    uchar3 color;
    TrilinearInterpolation(hash_table, blocks, vertex_pos, sdf, stats, color, converter);
    float3 val = ValToRGB(stats.duration, 0, 100);
    mesh.vertex(ptr).color = make_float3(val.x, val.y, val.z);
  }

  return ptr;
}

__device__
inline int GetVertex(Voxel& voxel, uint& vertex_idx) {
  voxel.ResetMutexes();// ???

  // It is guaranteed to be non-negative
  return voxel.vertex_ptrs[vertex_idx];
}

__device__
void RefineMesh(short& prev_cube, short& curr_cube, float d[8], int is_noise_bit[8]) {
  float kTr = 0.0075;

  /// Step 1: temporal
  short temporal_diff = curr_cube ^ prev_cube;
  int dist = 0;
  while (temporal_diff) {
    temporal_diff &= (temporal_diff - 1);
    dist++;
  }
  if (dist > 3) return;

  /// Step 2: Spatially closest
  float min_dist = 1e10;
  int min_idx = -1;
  for (int i = 0; i < 6; ++i) {
    short spatial_diff = curr_cube ^ kRegularCubeIndices[i];
    short hamming_dist = 0;
    float euclid_dist;

    for (int j = 0; j < 8; ++j) {
      short mask = (1 << j);
      if (mask & spatial_diff) {
        hamming_dist++;
        euclid_dist += fabs(d[j]);
        if (hamming_dist > 3) break;
      }
    }

    if (hamming_dist <= 3 && euclid_dist < min_dist) {
      min_dist = euclid_dist;
      min_idx = i;
    }
  }
  if (min_idx < 0) return;

  /// Step 3: Valid?
  int noise_bit[3];
  short hamming_dist = 0;
  short binary_xor = curr_cube ^ kRegularCubeIndices[min_idx];
  for (int j = 0; j < 8; ++j) {
    short mask = (1 << j);
    if (mask & binary_xor) {
      noise_bit[hamming_dist] = j;
      hamming_dist++;
    }
  }

  for (int j = 0; j < hamming_dist; ++j) {
    if (fabs(d[noise_bit[j]]) > kTr) {
      return;
    }
  }

  for (int i = 0; i < 8; ++i) {
    is_noise_bit[i] = 0;
  }
  for (int j = 0; j < hamming_dist; ++j) {
    //d[noise_bit[j]] = - d[noise_bit[j]];
    is_noise_bit[noise_bit[j]] = 1;
  }
  curr_cube = kRegularCubeIndices[min_idx];
}

__global__
void MarchingCubesPass1Kernel(
    HashTable        hash_table,
    EntryArray candidate_entries,
    BlockArray           blocks,
    Mesh             mesh,
    bool                use_fine_gradient,
    CoordinateConverter converter) {

  const HashEntry &entry = candidate_entries[blockIdx.x];
  const uint local_idx   = threadIdx.x;

  int3  voxel_base_pos  = converter.BlockToVoxel(entry.pos);
  uint3 voxel_local_pos = converter.IdxToVoxelLocalPos(local_idx);
  int3 voxel_pos        = voxel_base_pos + make_int3(voxel_local_pos);
  float3 world_pos      = converter.VoxelToWorld(voxel_pos);

  Voxel &this_voxel = blocks[entry.ptr].voxels[local_idx];

  //////////
  /// 1. Read the scalar values, see mc_tables.h
  const int   kVertexCount = 8;
  const float kVoxelSize   = converter.voxel_size;
  const float kThreshold   = 0.2f;
  const float kIsoLevel    = 0;

  float  d[kVertexCount];
  float3 p[kVertexCount];

  short cube_index = 0;
  this_voxel.prev_cube_idx = this_voxel.curr_cube_idx;
  this_voxel.curr_cube_idx = 0;

  /// Check 8 corners of a cube: are they valid?
  for (int i = 0; i < kVertexCount; ++i) {
    uint3 offset = make_uint3(kVtxOffset[i]);
    float weight;

    d[i] = GetSDF(hash_table, blocks, entry, voxel_local_pos + offset, weight, converter);
    if (weight < 20)
      return;

    if (fabs(d[i]) > kThreshold) return;

    if (d[i] < kIsoLevel) cube_index |= (1 << i);
    p[i] = world_pos + kVoxelSize * make_float3(offset);
  }
  this_voxel.curr_cube_idx = cube_index;
  if (cube_index == 0 || cube_index == 255) return;

  //int is_noise_bit[8];
  //RefineMesh(this_voxel.prev_cube_idx, this_voxel.curr_cube_idx, d, is_noise_bit);
  //cube_index = this_voxel.curr_cube_idx;

  const int kEdgeCount = 12;
#pragma unroll 1
  for (int i = 0; i < kEdgeCount; ++i) {
    if (kEdgeTable[cube_index] & (1 << i)) {
      int2  v_idx = kEdgeVertexTable[i];
      uint4 c_idx = kEdgeCubeTable[i];

      // Special noise-bit interpolation here: extrapolation
      float3 vertex_pos;
      vertex_pos = VertexIntersection(p[v_idx.x], p[v_idx.y],
                                      d[v_idx.x], d[v_idx.y], kIsoLevel);

      Voxel &voxel = GetVoxelRef(hash_table, blocks, entry,
                                voxel_local_pos + make_uint3(c_idx.x, c_idx.y, c_idx.z), converter);
      AllocateVertexWithMutex(hash_table, blocks, mesh,
                              voxel, c_idx.w, vertex_pos,
                              use_fine_gradient, converter);
    }
  }
}

__global__
void MarchingCubesPass2Kernel(
    HashTable        hash_table,
    EntryArray candidate_entries,
    BlockArray          blocks,
    Mesh             mesh,
    bool                use_fine_gradient,
    CoordinateConverter converter) {


  const HashEntry &entry = candidate_entries[blockIdx.x];
  const uint local_idx   = threadIdx.x;

  int3  voxel_base_pos  = converter.BlockToVoxel(entry.pos);
  uint3 voxel_local_pos = converter.IdxToVoxelLocalPos(local_idx);
  int3 voxel_pos        = voxel_base_pos + make_int3(voxel_local_pos);
  float3 world_pos      = converter.VoxelToWorld(voxel_pos);

  Voxel &this_voxel = blocks[entry.ptr].voxels[local_idx];

  /// Cube type unchanged: NO need to update triangles
//  if (this_cube.curr_cube_idx == this_cube.prev_cube_idx) {
//    blocks[entry.ptr].voxels[local_idx].stats.duration += 1.0f;
//    return;
//  }
//  blocks[entry.ptr].voxels[local_idx].stats.duration = 0;

  if (this_voxel.curr_cube_idx == 0 || this_voxel.curr_cube_idx == 255) {
    return;
  }

  //////////
  /// 2. Determine vertices (ptr allocated via (shared) edges
  /// If the program reach here, the voxels holding edges must exist
  /// This operation is in 2-pass
  /// pass2: Assign
  const int kEdgeCount = 12;
  int vertex_ptr[kEdgeCount];

#pragma unroll 1
  for (int i = 0; i < kEdgeCount; ++i) {
    if (kEdgeTable[this_voxel.curr_cube_idx] & (1 << i)) {
      uint4 c_idx = kEdgeCubeTable[i];
      uint3 voxel_p = voxel_local_pos + make_uint3(c_idx.x, c_idx.y, c_idx.z);
      Voxel &voxel = GetVoxelRef(hash_table, blocks, entry, voxel_p, converter);
      vertex_ptr[i] = GetVertex(voxel, c_idx.w);
    }
  }

  //////////
  /// 3. Assign triangles
  int i = 0;
  for (int t = 0;
       kTriangleTable[this_voxel.curr_cube_idx][t] != -1;
       t += 3, ++i) {
    int triangle_ptr = this_voxel.triangle_ptrs[i];
    if (triangle_ptr == FREE_PTR) {
      triangle_ptr = mesh.AllocTriangle();
    } else {
      mesh.ReleaseTriangle(mesh.triangle(triangle_ptr));
    }
    this_voxel.triangle_ptrs[i] = triangle_ptr;

    mesh.AssignTriangle(mesh.triangle(triangle_ptr),
                        make_int3(vertex_ptr[kTriangleTable[this_voxel.curr_cube_idx][t + 0]],
                                  vertex_ptr[kTriangleTable[this_voxel.curr_cube_idx][t + 1]],
                                  vertex_ptr[kTriangleTable[this_voxel.curr_cube_idx][t + 2]]));
    if (! use_fine_gradient) {
      mesh.ComputeTriangleNormal(mesh.triangle(triangle_ptr));
    }
  }
}

/// Garbage collection (ref count)
__global__
void RecycleTrianglesKernel(
    EntryArray candidate_entries,
    BlockArray       blocks,
    Mesh             mesh) {
  const HashEntry &entry = candidate_entries[blockIdx.x];

  const uint local_idx = threadIdx.x;  //inside an SDF block
  Voxel &voxel = blocks[entry.ptr].voxels[local_idx];

  int i = 0;
  for (int t = 0; kTriangleTable[voxel.curr_cube_idx][t] != -1; t += 3, ++i);

  for (; i < N_TRIANGLE; ++i) {
    int triangle_ptr = voxel.triangle_ptrs[i];
    if (triangle_ptr == FREE_PTR) continue;

    // Clear ref_count of its pointed vertices
    mesh.ReleaseTriangle(mesh.triangle(triangle_ptr));
    mesh.triangle(triangle_ptr).Clear();
    mesh.FreeTriangle(triangle_ptr);
    voxel.triangle_ptrs[i] = FREE_PTR;
  }
}

__global__
void RecycleVerticesKernel(
    EntryArray candidate_entries,
    BlockArray           blocks,
    Mesh             mesh) {
  const HashEntry &entry = candidate_entries[blockIdx.x];
  const uint local_idx = threadIdx.x;

  Voxel &voxel = blocks[entry.ptr].voxels[local_idx];

#pragma unroll 1
  for (int i = 0; i < 3; ++i) {
    if (voxel.vertex_ptrs[i] != FREE_PTR &&
        mesh.vertex(voxel.vertex_ptrs[i]).ref_count == 0) {
      mesh.vertex(voxel.vertex_ptrs[i]).Clear();
      mesh.FreeVertex(voxel.vertex_ptrs[i]);
      voxel.vertex_ptrs[i] = FREE_PTR;
    }
  }
}

/// Only update Laplacian at current
#ifdef STATS
__global__
void UpdateStatisticsKernel(HashTable        hash_table,
                            EntryArray candidate_entries,
                            BlockArray           blocks) {

  const HashEntry &entry = candidate_entries.entries[blockIdx.x];
  const uint local_idx   = threadIdx.x;

  int3  voxel_base_pos  = BlockToVoxel(entry.pos);
  uint3 voxel_local_pos = IdxToVoxelLocalPos(local_idx);
  int3 voxel_pos        = voxel_base_pos + make_int3(voxel_local_pos);

  const int3 offset[] = {
      make_int3(1, 0, 0),
      make_int3(-1, 0, 0),
      make_int3(0, 1, 0),
      make_int3(0, -1, 0),
      make_int3(0, 0, 1),
      make_int3(0, 0, -1)
  };

  float sdf = blocks[entry.ptr].voxels[local_idx].sdf;
  float laplacian = 8 * sdf;

  for (int i = 0; i < 3; ++i) {
    Voxel vp = GetVoxel(hash_table, blocks, VoxelToWorld(voxel_pos + offset[2*i]));
    Voxel vn = GetVoxel(hash_table, blocks, VoxelToWorld(voxel_pos + offset[2*i+1]));
    if (vp.weight == 0 || vn.weight == 0) {
      blocks[entry.ptr].voxels[local_idx].stats.laplacian = 1;
      return;
    }
    laplacian += vp.sdf + vn.sdf;
  }

  blocks[entry.ptr].voxels[local_idx].stats.laplacian = laplacian;
}
#endif

////////////////////
/// Host code
////////////////////
void MarchingCubes(EntryArray& candidate_entries,
                   HashTable& hash_table,
                   BlockArray& blocks,
                   Mesh& mesh,
                   bool use_fine_gradient,
                   CoordinateConverter& converter) {
  uint occupied_block_count = candidate_entries.count();
  LOG(INFO) << "Marching cubes block count: " << occupied_block_count;
  if (occupied_block_count <= 0)
    return;

  const uint threads_per_block = BLOCK_SIZE;
  const dim3 grid_size(occupied_block_count, 1);
  const dim3 block_size(threads_per_block, 1);

  /// First update statistics
#ifdef STATS
  UpdateStatisticsKernel<<<grid_size, block_size>>>(
      hash_table,
          candidate_entries,
          blocks);
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipGetLastError());
#endif

  /// Use divide and conquer to avoid read-write conflict
  Timer timer;
  timer.Tick();
  MarchingCubesPass1Kernel<<<grid_size, block_size>>>(
      hash_table,
          candidate_entries,
          blocks,
          mesh,
          use_fine_gradient,
          converter);
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipGetLastError());
  double pass1_seconds = timer.Tock();
  LOG(INFO) << "Pass1 duration: " << pass1_seconds;

  timer.Tick();
  MarchingCubesPass2Kernel<<<grid_size, block_size>>>(
      hash_table,
          candidate_entries,
          blocks,
          mesh,
          use_fine_gradient,
          converter);
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipGetLastError());
  double pass2_seconds = timer.Tock();
  LOG(INFO) << "Pass2 duration: " << pass2_seconds;

  RecycleTrianglesKernel<<<grid_size, block_size>>>(candidate_entries, blocks, mesh);
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipGetLastError());

  RecycleVerticesKernel<<<grid_size, block_size>>>(candidate_entries, blocks, mesh);
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipGetLastError());
}
