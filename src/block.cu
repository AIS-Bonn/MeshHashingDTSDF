#include "hip/hip_runtime.h"
#include "block.h"
#include <hip/hip_runtime_api.h>

__global__
void ResetBlocksKernel(VoxelBlocksGPU blocks, int block_count) {
  uint idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < block_count) {
    blocks[idx].Clear();
  }
}

void VoxelBlocks::Alloc(uint block_count) {
  checkCudaErrors(hipMalloc(&gpu_data_,
                             sizeof(VoxelBlock) * block_count));
}

void VoxelBlocks::Free() {
  checkCudaErrors(hipFree(gpu_data_));

}

VoxelBlocks::VoxelBlocks() {
  block_count_ = 0;
}

VoxelBlocks::~VoxelBlocks() {
  Free();
}

void VoxelBlocks::Reset() {
  const int threads_per_block = 64;

  if (block_count_ == 0) return;
  const dim3 grid_size((block_count_ + threads_per_block - 1)
                       / threads_per_block, 1);
  const dim3 block_size(threads_per_block, 1);

  ResetBlocksKernel<<<grid_size, block_size>>>(gpu_data_, block_count_);
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipGetLastError());
}

void VoxelBlocks::Resize(uint block_count) {
  block_count_ = block_count;
  Alloc(block_count);
}