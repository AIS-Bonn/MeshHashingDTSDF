#include "hip/hip_runtime.h"
#include "block.h"
#include <hip/hip_runtime_api.h>
#include <>

////////////////////
/// class Blocks
////////////////////

////////////////////
/// Device code
////////////////////
__global__
void ResetBlocksKernel(BlocksGPU blocks, int block_count) {
  const uint idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < block_count) {
    blocks[idx].Clear();
  }
}

////////////////////
/// Host code
////////////////////

/// Life cycle
Blocks::Blocks() {
  block_count_  = 0;
}

Blocks::~Blocks() {
  Free();
}

void Blocks::Alloc(uint block_count) {
  checkCudaErrors(hipMalloc(&gpu_data_, sizeof(Block) * block_count));
}

void Blocks::Free() {
  checkCudaErrors(hipFree(gpu_data_));
}

void Blocks::Resize(uint block_count) {
  block_count_ = block_count;
  Alloc(block_count);
  Reset();
}

/// Reset
void Blocks::Reset() {
  const int threads_per_block = 64;

  if (block_count_ == 0) return;
  const dim3 grid_size((block_count_ + threads_per_block - 1)
                      / threads_per_block, 1);
  const dim3 block_size(threads_per_block, 1);

  ResetBlocksKernel<<<grid_size, block_size>>>(gpu_data_, block_count_);
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipGetLastError());
}