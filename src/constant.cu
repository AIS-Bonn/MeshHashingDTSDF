#include "params.h"

#include <hip/hip_runtime_api.h>

__constant__ SDFParams kSDFParams;
void SetConstantSDFParams(const SDFParams& params) {
  size_t size;
  checkCudaErrors(hipGetSymbolSize(&size, HIP_SYMBOL(kSDFParams)));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(kSDFParams), &params,
                                     size, 0,
                                     hipMemcpyHostToDevice));
}