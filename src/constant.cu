#include "params.h"

#include <hip/hip_runtime_api.h>

__constant__ SDFParams kSDFParams;
void SetConstantSDFParams(const SDFParams& params) {
  size_t size;
  checkCudaErrors(hipGetSymbolSize(&size, HIP_SYMBOL(kSDFParams)));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(kSDFParams), &params, size, 0, hipMemcpyHostToDevice));
}

__constant__ SensorParams kSensorParams;
void SetConstantSensorParams(const SensorParams& params) {
  size_t size;
  checkCudaErrors(hipGetSymbolSize(&size, HIP_SYMBOL(kSensorParams)));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(kSensorParams), &params, size, 0, hipMemcpyHostToDevice));
}

__constant__ RayCasterParams kRayCasterParams;
void SetConstantRayCasterParams(const RayCasterParams& params) {
  size_t size;
  checkCudaErrors(hipGetSymbolSize(&size, HIP_SYMBOL(kRayCasterParams)));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(kRayCasterParams), &params, size, 0, hipMemcpyHostToDevice));
}