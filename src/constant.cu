#include "hash_param.h"
#include "ray_caster_param.h"
#include "sensor_param.h"
#include <hip/hip_runtime_api.h>

__constant__ HashParams kHashParams;
void SetConstantHashParams(const HashParams& params) {
  size_t size;
  checkCudaErrors(hipGetSymbolSize(&size, HIP_SYMBOL(kHashParams)));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(kHashParams), &params, size, 0, hipMemcpyHostToDevice));
}

__constant__ SensorParams kSensorParams;
void SetConstantSensorParams(const SensorParams& params) {
  size_t size;
  checkCudaErrors(hipGetSymbolSize(&size, HIP_SYMBOL(kSensorParams)));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(kSensorParams), &params, size, 0, hipMemcpyHostToDevice));
}

__constant__ RayCasterParams kRayCasterParams;
void SetConstantRayCasterParams(const RayCasterParams& params) {
  size_t size;
  checkCudaErrors(hipGetSymbolSize(&size, HIP_SYMBOL(kRayCasterParams)));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(kRayCasterParams), &params, size, 0, hipMemcpyHostToDevice));
}