#include "hash_param.h"
#include "ray_caster_param.h"
#include "sensor_param.h"
#include <hip/hip_runtime_api.h>

__constant__ HashParams c_hashParams;
void updateConstantHashParams(const HashParams& params) {
  size_t size;
  checkCudaErrors(hipGetSymbolSize(&size, HIP_SYMBOL(c_hashParams)));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_hashParams), &params, size, 0, hipMemcpyHostToDevice));
}

__constant__ DepthCameraParams c_depthCameraParams;
void updateConstantDepthCameraParams(const DepthCameraParams& params) {
  size_t size;
  checkCudaErrors(hipGetSymbolSize(&size, HIP_SYMBOL(c_depthCameraParams)));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_depthCameraParams), &params, size, 0, hipMemcpyHostToDevice));
}

__constant__ RayCastParams c_rayCastParams;
void updateConstantRayCastParams(const RayCastParams& params) {
  size_t size;
  checkCudaErrors(hipGetSymbolSize(&size, HIP_SYMBOL(c_rayCastParams)));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_rayCastParams), &params, size, 0, hipMemcpyHostToDevice));
}