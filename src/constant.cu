#include "params.h"

#include <hip/hip_runtime_api.h>

__device__ __constant__ SDFParams kSDFParams;
void SetConstantSDFParams(const SDFParams& params) {
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(kSDFParams), &params,
                                     sizeof(SDFParams), 0,
                                     hipMemcpyHostToDevice));
}