#include "hash_param.h"
#include "ray_caster_param.h"
#include "sensor_param.h"
#include <hip/hip_runtime_api.h>

__constant__ HashParams kHashParams;
void UpdateConstantHashParams(const HashParams& params) {
  size_t size;
  checkCudaErrors(hipGetSymbolSize(&size, HIP_SYMBOL(kHashParams)));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(kHashParams), &params, size, 0, hipMemcpyHostToDevice));
}

__constant__ DepthCameraParams c_depthCameraParams;
void updateConstantDepthCameraParams(const DepthCameraParams& params) {
  size_t size;
  checkCudaErrors(hipGetSymbolSize(&size, HIP_SYMBOL(c_depthCameraParams)));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_depthCameraParams), &params, size, 0, hipMemcpyHostToDevice));
}

__constant__ RayCastParams c_rayCastParams;
void updateConstantRayCastParams(const RayCastParams& params) {
  size_t size;
  checkCudaErrors(hipGetSymbolSize(&size, HIP_SYMBOL(c_rayCastParams)));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_rayCastParams), &params, size, 0, hipMemcpyHostToDevice));
}