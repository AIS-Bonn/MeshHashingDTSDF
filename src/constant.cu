#include "hash_param.h"
#include "ray_caster_param.h"
#include "sensor_param.h"
#include <hip/hip_runtime_api.h>

__constant__ HashParams kHashParams;
void UpdateConstantHashParams(const HashParams& params) {
  size_t size;
  checkCudaErrors(hipGetSymbolSize(&size, HIP_SYMBOL(kHashParams)));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(kHashParams), &params, size, 0, hipMemcpyHostToDevice));
}

__constant__ SensorParams kSensorParams;
void UpdateConstantSensorParams(const SensorParams& params) {
  size_t size;
  checkCudaErrors(hipGetSymbolSize(&size, HIP_SYMBOL(kSensorParams)));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(kSensorParams), &params, size, 0, hipMemcpyHostToDevice));
}

__constant__ RayCastParams kRayCastParams;
void UpdateConstantRayCastParams(const RayCastParams& params) {
  size_t size;
  checkCudaErrors(hipGetSymbolSize(&size, HIP_SYMBOL(kRayCastParams)));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(kRayCastParams), &params, size, 0, hipMemcpyHostToDevice));
}