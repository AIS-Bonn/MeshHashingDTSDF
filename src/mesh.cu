#include "hip/hip_runtime.h"
#include "mesh.h"

#include <hip/hip_runtime_api.h>
#include <>

////////////////////
/// class Mesh
////////////////////

////////////////////
/// Device code
////////////////////
__global__
void ResetHeapKernel(MeshGPU mesh) {
  const uint idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < kMaxVertexCount) {
    mesh.vertex_heap[idx] = kMaxVertexCount - idx - 1;
    mesh.triangle_heap[idx] = kMaxVertexCount - idx - 1;
    mesh.vertices[idx].Clear();
    mesh.triangles[idx].Clear();
  }
}

////////////////////
/// Host code
////////////////////
Mesh::Mesh() {}

Mesh::~Mesh() {
  Free();
}

void Mesh::Alloc(uint vertex_count, uint triangle_count) {
  checkCudaErrors(hipMalloc(&gpu_data_.vertex_heap,
                             sizeof(uint) * kMaxVertexCount));
  checkCudaErrors(hipMalloc(&gpu_data_.vertex_heap_counter, sizeof(uint)));
  checkCudaErrors(hipMalloc(&gpu_data_.vertices,
                             sizeof(Vertex) * kMaxVertexCount));

  checkCudaErrors(hipMalloc(&gpu_data_.triangle_heap,
                             sizeof(uint) * kMaxVertexCount));
  checkCudaErrors(hipMalloc(&gpu_data_.triangle_heap_counter, sizeof(uint)));
  checkCudaErrors(hipMalloc(&gpu_data_.triangles,
                             sizeof(Triangle) * kMaxVertexCount));
}

void Mesh::Free() {
  checkCudaErrors(hipFree(gpu_data_.vertex_heap));
  checkCudaErrors(hipFree(gpu_data_.vertex_heap_counter));
  checkCudaErrors(hipFree(gpu_data_.vertices));

  checkCudaErrors(hipFree(gpu_data_.triangle_heap));
  checkCudaErrors(hipFree(gpu_data_.triangle_heap_counter));
  checkCudaErrors(hipFree(gpu_data_.triangles));
}

void Mesh::Resize(uint vertex_count, uint triangle_count) {
  Alloc(vertex_count, triangle_count);
  Reset();
}

void Mesh::Reset() {
  uint val = kMaxVertexCount - 1;
  checkCudaErrors(hipMemcpy(gpu_data_.vertex_heap_counter, &val,
                             sizeof(uint),
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(gpu_data_.triangle_heap_counter, &val,
                             sizeof(uint),
                             hipMemcpyHostToDevice));

  const int threads_per_block = 64;
  const dim3 grid_size((kMaxVertexCount + threads_per_block - 1)
                       / threads_per_block, 1);
  const dim3 block_size(threads_per_block, 1);

  ResetHeapKernel<<<grid_size, block_size>>>(gpu_data_);
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipGetLastError());
}

////////////////////
/// class CompactMesh
////////////////////

/// Life cycle
CompactMesh::CompactMesh() {}

CompactMesh::~CompactMesh() {
  Free();
}

void CompactMesh::Alloc(uint vertex_count, uint triangle_count) {
  checkCudaErrors(hipMalloc(&gpu_data_.vertex_index_remapper,
                             sizeof(int) * kMaxVertexCount));

  checkCudaErrors(hipMalloc(&gpu_data_.vertex_counter,
                             sizeof(uint)));
  checkCudaErrors(hipMalloc(&gpu_data_.vertices_ref_count,
                             sizeof(int) * kMaxVertexCount));
  checkCudaErrors(hipMalloc(&gpu_data_.vertices,
                             sizeof(float3) * kMaxVertexCount));

  checkCudaErrors(hipMalloc(&gpu_data_.triangle_counter,
                             sizeof(uint)));
  checkCudaErrors(hipMalloc(&gpu_data_.triangles_ref_count,
                             sizeof(int) * kMaxVertexCount));
  checkCudaErrors(hipMalloc(&gpu_data_.triangles,
                             sizeof(int3) * kMaxVertexCount));
}

void CompactMesh::Free() {
  checkCudaErrors(hipFree(gpu_data_.vertex_index_remapper));

  checkCudaErrors(hipFree(gpu_data_.vertex_counter));
  checkCudaErrors(hipFree(gpu_data_.vertices_ref_count));
  checkCudaErrors(hipFree(gpu_data_.vertices));

  checkCudaErrors(hipFree(gpu_data_.triangle_counter));
  checkCudaErrors(hipFree(gpu_data_.triangles_ref_count));
  checkCudaErrors(hipFree(gpu_data_.triangles));
}

void CompactMesh::Resize(uint vertex_count, uint triangle_count) {
  Alloc(vertex_count, triangle_count);
  Reset();
}

/// Reset
void CompactMesh::Reset() {
  checkCudaErrors(hipMemset(gpu_data_.vertex_index_remapper, 0xff,
                             sizeof(int) * kMaxVertexCount));
  checkCudaErrors(hipMemset(gpu_data_.vertices_ref_count, 0,
                             sizeof(int) * kMaxVertexCount));
  checkCudaErrors(hipMemset(gpu_data_.vertex_counter,
                             0, sizeof(uint)));
  checkCudaErrors(hipMemset(gpu_data_.triangles_ref_count, 0,
                             sizeof(int) * kMaxVertexCount));
  checkCudaErrors(hipMemset(gpu_data_.triangle_counter,
                             0, sizeof(uint)));
}