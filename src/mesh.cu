#include "hip/hip_runtime.h"
#include <glog/logging.h>
#include "mesh.h"

#include "mc_tables.h"

__global__
void ResetHeapKernel(MeshData mesh_data) {
  const uint max_vertice_count = 10000000;
  uint idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx == 0) {
    mesh_data.vertex_heap_counter[0] = max_vertice_count - 1;
    mesh_data.triangle_heap_counter[0] = max_vertice_count - 1;
  }

  if (idx < max_vertice_count) {
    mesh_data.vertex_heap[idx] = max_vertice_count - idx - 1;
    mesh_data.triangle_heap[idx] = max_vertice_count - idx - 1;
    mesh_data.vertices[idx].Clear();
    mesh_data.triangles[idx].Clear();
  }
}

/// At current we suppose 1 - 1 correspondence for voxels
/// No deletion of vertices is considered
__device__
float3 VertexIntersection(const float3& p1, const float3 p2,
                          const float& v1,  const float& v2, const float& isolevel) {
  if (fabs(v1 - isolevel) < 0.00001) return p1;
  if (fabs(v2 - isolevel) < 0.00001) return p2;
  float mu = (isolevel - v1) / (v2 - v1);
  float3 p = make_float3(p1.x + mu * (p2.x - p1.x),
                         p1.y + mu * (p2.y - p1.y),
                         p1.z + mu * (p2.z - p1.z));
  return p;
}

__device__
inline Voxel GetVoxel(HashTableGPU<VoxelBlock>& scalar_table,
                      const HashEntry& curr_entry,
                      uint3 voxel_local_pos,
                      const uint3 local_offset) {
  Voxel v; v.Clear();

  voxel_local_pos = voxel_local_pos + local_offset;
  int3 block_offset = make_int3(voxel_local_pos.x / BLOCK_SIDE_LENGTH,
                                voxel_local_pos.y / BLOCK_SIDE_LENGTH,
                                voxel_local_pos.z / BLOCK_SIDE_LENGTH);
  if (block_offset.x == 0
      && block_offset.y == 0
      && block_offset.z == 0) {
    v = scalar_table.values[curr_entry.ptr](VoxelLocalPosToIdx(voxel_local_pos));
  } else {
    HashEntry entry = scalar_table.GetEntry(curr_entry.pos + block_offset);
    if (entry.ptr == FREE_ENTRY) return v;
    int3 voxel_local_pos_ = make_int3(voxel_local_pos.x % BLOCK_SIDE_LENGTH,
                                      voxel_local_pos.y % BLOCK_SIDE_LENGTH,
                                      voxel_local_pos.z % BLOCK_SIDE_LENGTH);
    int i = VoxelPosToIdx(voxel_local_pos_);
    v = scalar_table.values[entry.ptr](i);
  }

  return v;
}

__device__
inline MeshCube& GetMeshCube(HashTableGPU<MeshCubeBlock>& mesh_table,
                                     MeshData &mesh_data,
                                     const HashEntry& curr_entry,
                                     uint3 voxel_local_pos,
                                     const uint3 local_offset) {

  voxel_local_pos = voxel_local_pos + local_offset;
  int3 block_offset = make_int3(voxel_local_pos.x / BLOCK_SIDE_LENGTH,
                                voxel_local_pos.y / BLOCK_SIDE_LENGTH,
                                voxel_local_pos.z / BLOCK_SIDE_LENGTH);

  if (block_offset.x == 0
      && block_offset.y == 0
      && block_offset.z == 0) {
    return mesh_table.values[curr_entry.ptr](VoxelLocalPosToIdx(voxel_local_pos));
  } else {
    HashEntry entry = mesh_table.GetEntry(curr_entry.pos + block_offset);
    if (entry.ptr == FREE_ENTRY) {
      printf("Should never reach here! %d %d %d\n",
             voxel_local_pos.x,
             voxel_local_pos.y,
             voxel_local_pos.z);
    }
    /// this should never happen
    /// if (entry.ptr == FREE_ENTRY) return indices;
    int3 voxel_local_pos_ = make_int3(voxel_local_pos.x % BLOCK_SIDE_LENGTH,
                                      voxel_local_pos.y % BLOCK_SIDE_LENGTH,
                                      voxel_local_pos.z % BLOCK_SIDE_LENGTH);
    int i = VoxelPosToIdx(voxel_local_pos_);
    return mesh_table.values[entry.ptr](i);
  }
}

// TODO(wei): add locks
__global__
void MarchingCubesKernel(HashTableGPU<VoxelBlock> scalar_table,
                         HashTableGPU<MeshCubeBlock> mesh_table,
                         MeshData mesh_data) {
  const float isolevel = 0;

  const HashEntry &mesh_entry
          = mesh_table.compacted_hash_entries[blockIdx.x];
  const HashEntry &scalar_entry = scalar_table.GetEntry(mesh_entry.pos);
  if (scalar_entry.ptr == FREE_ENTRY) {
    /// correct for the 1st frame, incorrect for the 2nd
    return;
    //printf("MarchingCubesKernel: should never reach here!\n");
  }

  int3  voxel_base_pos = BlockToVoxel(scalar_entry.pos);
  const uint local_idx = threadIdx.x;  //inside of an SDF block
  uint3 voxel_local_pos = IdxToVoxelLocalPos(local_idx);
  // TODO(wei): deal with border condition alone to save processing time?

  int3 voxel_pos = voxel_base_pos + make_int3(voxel_local_pos);
  float3 world_pos = VoxelToWorld(voxel_pos);

  //////////
  /// 1. Read the scalar values
  /// Refer to paulbourke.net/geometry/polygonise
  /// Our coordinate system:
  ///       ^
  ///      /
  ///    z
  ///   /
  /// o -- x -->
  /// |
  /// y
  /// |
  /// v
  // 0 -> 011
  // 1 -> 111
  // 2 -> 110
  // 3 -> 010
  // 4 -> 001
  // 5 -> 101
  // 6 -> 100
  // 7 -> 000
  Voxel  v;
  float  d[8];
  float3 p[8];

  float voxel_size = kSDFParams.voxel_size;
  v = GetVoxel(scalar_table, scalar_entry, voxel_local_pos, make_uint3(0, 1, 1));
  if (v.weight == 0) return;
  p[0] = world_pos + voxel_size * make_float3(0, 1, 1);
  d[0] = v.sdf;

  v = GetVoxel(scalar_table, scalar_entry, voxel_local_pos, make_uint3(1, 1, 1));
  if (v.weight == 0) return;
  p[1] = world_pos + voxel_size * make_float3(1, 1, 1);
  d[1] = v.sdf;

  v = GetVoxel(scalar_table, scalar_entry, voxel_local_pos, make_uint3(1, 1, 0));
  if (v.weight == 0) return;
  p[2] = world_pos + voxel_size * make_float3(1, 1, 0);
  d[2] = v.sdf;

  v = GetVoxel(scalar_table, scalar_entry, voxel_local_pos, make_uint3(0, 1, 0));
  if (v.weight == 0) return;
  p[3] = world_pos + voxel_size * make_float3(0, 1, 0);
  d[3] = v.sdf;

  v = GetVoxel(scalar_table, scalar_entry, voxel_local_pos, make_uint3(0, 0, 1));
  if (v.weight == 0) return;
  p[4] = world_pos + voxel_size * make_float3(0, 0, 1);
  d[4] = v.sdf;

  v = GetVoxel(scalar_table, scalar_entry, voxel_local_pos, make_uint3(1, 0, 1));
  if (v.weight == 0) return;
  p[5] = world_pos + voxel_size * make_float3(1, 0, 1);
  d[5] = v.sdf;

  v = GetVoxel(scalar_table, scalar_entry, voxel_local_pos, make_uint3(1, 0, 0));
  if (v.weight == 0) return;
  p[6] = world_pos + voxel_size * make_float3(1, 0, 0);
  d[6] = v.sdf;

  v = GetVoxel(scalar_table, scalar_entry, voxel_local_pos, make_uint3(0, 0, 0));
  if (v.weight == 0) return;
  p[7] = world_pos + voxel_size * make_float3(0, 0, 0);
  d[7] = v.sdf;

  //////////
  /// 2. Determine cube type
  int cube_index = 0;
  if (d[0] < isolevel) cube_index |= 1;
  if (d[1] < isolevel) cube_index |= 2;
  if (d[2] < isolevel) cube_index |= 4;
  if (d[3] < isolevel) cube_index |= 8;
  if (d[4] < isolevel) cube_index |= 16;
  if (d[5] < isolevel) cube_index |= 32;
  if (d[6] < isolevel) cube_index |= 64;
  if (d[7] < isolevel) cube_index |= 128;

  if (kEdgeTable[cube_index] == 0 || kEdgeTable[cube_index] == 255)
    return;

  //////////
  /// 3. Determine vertices (ptr allocated via (shared) edges
  /// If the program reach here, the voxels holding edges must exist
  // 0 -> 011.x, (0, 1)
  // 1 -> 110.z, (1, 2)
  // 2 -> 010.x, (2, 3)
  // 3 -> 010.z, (3, 0)
  // 4 -> 001.x, (4, 5)
  // 5 -> 100.z, (5, 6)
  // 6 -> 000.x, (6, 7)
  // 7 -> 000.z, (7, 4)
  // 8 -> 001.y, (4, 0)
  // 9 -> 101.y, (5, 1)
  //10 -> 100.y, (6, 2)
  //11 -> 000.y, (7, 3)
  int vertex_ptr[12];
  float3 vertex_pos;
  int ptr;
  /// plane y = 1
  if (kEdgeTable[cube_index] & 1) {
    vertex_pos = VertexIntersection(p[0], p[1], d[0], d[1], isolevel);

    MeshCube& cube = GetMeshCube(mesh_table, mesh_data, mesh_entry, voxel_local_pos, make_uint3(0, 1, 1));
    ptr = cube.vertex_ptrs.x;
    if (ptr == -1) ptr = mesh_data.AllocVertexHeap();
    mesh_data.vertices[ptr].pos = vertex_pos;
    cube.vertex_ptrs.x = ptr;
    vertex_ptr[0] = ptr;
  }
  if (kEdgeTable[cube_index] & 2) {
    vertex_pos = VertexIntersection(p[1], p[2], d[1], d[2], isolevel);

    MeshCube &cube = GetMeshCube(mesh_table, mesh_data, mesh_entry, voxel_local_pos, make_uint3(1, 1, 0));
    ptr = cube.vertex_ptrs.z;
    if (ptr == -1) ptr = mesh_data.AllocVertexHeap();
    mesh_data.vertices[ptr].pos = vertex_pos;
    cube.vertex_ptrs.z = ptr;
    vertex_ptr[1] = ptr;
  }
  if (kEdgeTable[cube_index] & 4) {
    vertex_pos = VertexIntersection(p[2], p[3], d[2], d[3], isolevel);

    MeshCube &cube = GetMeshCube(mesh_table, mesh_data, mesh_entry, voxel_local_pos, make_uint3(0, 1, 0));
    ptr = cube.vertex_ptrs.x;
    if (ptr == -1) ptr = mesh_data.AllocVertexHeap();
    mesh_data.vertices[ptr].pos = vertex_pos;
    cube.vertex_ptrs.x = ptr;
    vertex_ptr[2] = ptr;
  }
  if (kEdgeTable[cube_index] & 8) {
    vertex_pos = VertexIntersection(p[3], p[0], d[3], d[0], isolevel);

    MeshCube& cube = GetMeshCube(mesh_table, mesh_data, mesh_entry, voxel_local_pos, make_uint3(0, 1, 0));
    ptr = cube.vertex_ptrs.z;
    if (ptr == -1) ptr = mesh_data.AllocVertexHeap();
    mesh_data.vertices[ptr].pos = vertex_pos;
    cube.vertex_ptrs.z = ptr;
    vertex_ptr[3] = ptr;
  }

  /// plane y = 0
  if (kEdgeTable[cube_index] & 16) {
    vertex_pos = VertexIntersection(p[4], p[5], d[4], d[5], isolevel);

    MeshCube& cube = GetMeshCube(mesh_table, mesh_data, mesh_entry, voxel_local_pos, make_uint3(0, 0, 1));
    ptr = cube.vertex_ptrs.x;
    if (ptr == -1) ptr = mesh_data.AllocVertexHeap();
    mesh_data.vertices[ptr].pos = vertex_pos;
    cube.vertex_ptrs.x = ptr;
    vertex_ptr[4] = ptr;
  }
  if (kEdgeTable[cube_index] & 32) {
    vertex_pos = VertexIntersection(p[5], p[6], d[5], d[6], isolevel);

    MeshCube& cube = GetMeshCube(mesh_table, mesh_data, mesh_entry, voxel_local_pos, make_uint3(1, 0, 0));
    ptr = cube.vertex_ptrs.z;
    if (ptr == -1) ptr = mesh_data.AllocVertexHeap();
    mesh_data.vertices[ptr].pos = vertex_pos;
    cube.vertex_ptrs.z = ptr;
    vertex_ptr[5] = ptr;
  }
  if (kEdgeTable[cube_index] & 64) {
    vertex_pos = VertexIntersection(p[6], p[7], d[6], d[7], isolevel);

    MeshCube& cube = GetMeshCube(mesh_table, mesh_data, mesh_entry, voxel_local_pos, make_uint3(0, 0, 0));
    ptr = cube.vertex_ptrs.x;
    if (ptr == -1) ptr = mesh_data.AllocVertexHeap();
    mesh_data.vertices[ptr].pos = vertex_pos;
    cube.vertex_ptrs.x = ptr;
    vertex_ptr[6] = ptr;
  }
  if (kEdgeTable[cube_index] & 128) {
    vertex_pos = VertexIntersection(p[7], p[4], d[7], d[4], isolevel);

    MeshCube& cube = GetMeshCube(mesh_table, mesh_data, mesh_entry, voxel_local_pos, make_uint3(0, 0, 0));
    ptr = cube.vertex_ptrs.z;
    if (ptr == -1) ptr = mesh_data.AllocVertexHeap();
    mesh_data.vertices[ptr].pos = vertex_pos;
    cube.vertex_ptrs.z = ptr;
    vertex_ptr[7] = ptr;
  }

  /// vertical
  if (kEdgeTable[cube_index] & 256) {
    vertex_pos = VertexIntersection(p[4], p[0], d[4], d[0], isolevel);

    MeshCube& cube = GetMeshCube(mesh_table, mesh_data, mesh_entry, voxel_local_pos, make_uint3(0, 0, 1));
    ptr = cube.vertex_ptrs.y;
    if (ptr == -1) ptr = mesh_data.AllocVertexHeap();
    mesh_data.vertices[ptr].pos = vertex_pos;
    cube.vertex_ptrs.y = ptr;
    vertex_ptr[8] = ptr;
  }
  if (kEdgeTable[cube_index] & 512) {
    vertex_pos = VertexIntersection(p[5], p[1], d[5], d[1], isolevel);

    MeshCube &cube = GetMeshCube(mesh_table, mesh_data, mesh_entry, voxel_local_pos, make_uint3(1, 0, 1));
    ptr = cube.vertex_ptrs.y;
    if (ptr == -1) ptr = mesh_data.AllocVertexHeap();
    mesh_data.vertices[ptr].pos = vertex_pos;
    cube.vertex_ptrs.y = ptr;
    vertex_ptr[9] = ptr;
  }
  if (kEdgeTable[cube_index] & 1024) {
    vertex_pos = VertexIntersection(p[6], p[2], d[6], d[2], isolevel);

    MeshCube &cube = GetMeshCube(mesh_table, mesh_data, mesh_entry, voxel_local_pos, make_uint3(1, 0, 0));
    ptr = cube.vertex_ptrs.y;
    if (ptr == -1) ptr = mesh_data.AllocVertexHeap();
    mesh_data.vertices[ptr].pos = vertex_pos;
    cube.vertex_ptrs.y = ptr;
    vertex_ptr[10] = ptr;
  }
  if (kEdgeTable[cube_index] & 2048) {
    vertex_pos = VertexIntersection(p[7], p[3], d[7], d[3], isolevel);

    MeshCube &cube = GetMeshCube(mesh_table, mesh_data, mesh_entry, voxel_local_pos, make_uint3(0, 0, 0));
    ptr = cube.vertex_ptrs.y;
    if (ptr == -1) ptr = mesh_data.AllocVertexHeap();
    mesh_data.vertices[ptr].pos = vertex_pos;
    cube.vertex_ptrs.y = ptr;
    vertex_ptr[11] = ptr;
  }

  MeshCube &cube = mesh_table.values[mesh_entry.ptr](local_idx);
  cube.cube_index = cube_index;
  int i = 0;
  for (int t = 0; kTriangleTable[cube_index][t] != -1; t += 3, ++i) {
    int triangle_ptr = cube.triangle_ptr[i];
    if (triangle_ptr == -1)
      triangle_ptr = mesh_data.AllocTriangleHeap();

    cube.triangle_ptr[i] = triangle_ptr;

    Triangle triangle; triangle.Clear();
    triangle.vertex_ptrs.x = vertex_ptr[kTriangleTable[cube_index][t + 0]];
    triangle.vertex_ptrs.y = vertex_ptr[kTriangleTable[cube_index][t + 1]];
    triangle.vertex_ptrs.z = vertex_ptr[kTriangleTable[cube_index][t + 2]];

    mesh_data.triangles[triangle_ptr] = triangle;
  }

}

__global__
void GarbageCollectionKernel(HashTableGPU<MeshCubeBlock> mesh_table, MeshData mesh_data) {
  const HashEntry &mesh_entry = mesh_table.compacted_hash_entries[blockIdx.x];

  const uint local_idx = threadIdx.x;  //inside of an SDF block
  MeshCube &cube = mesh_table.values[mesh_entry.ptr](local_idx);

  int t = 0;
  for (; kTriangleTable[cube.cube_index][t] != -1; t += 3);

  for (; t < 5; ++t) {
    int triangle_ptr = cube.triangle_ptr[t];
    if (triangle_ptr == -1) continue;
    cube.triangle_ptr[t] = -1;
    mesh_data.triangles[triangle_ptr].Clear();
    mesh_data.FreeTriangleHeap(triangle_ptr);
  }
}

Mesh::Mesh(const HashParams &params) {
  checkCudaErrors(hipMalloc(&mesh_data_.vertex_heap,
                             sizeof(uint) * kMaxVertexCount));
  checkCudaErrors(hipMalloc(&mesh_data_.vertex_heap_counter, sizeof(uint)));
  checkCudaErrors(hipMalloc(&mesh_data_.vertices,
                             sizeof(Vertex) * kMaxVertexCount));
  checkCudaErrors(hipMalloc(&mesh_data_.triangle_heap,
                             sizeof(uint) * kMaxVertexCount));
  checkCudaErrors(hipMalloc(&mesh_data_.triangle_heap_counter, sizeof(uint)));
  checkCudaErrors(hipMalloc(&mesh_data_.triangles,
                             sizeof(Triangle) * kMaxVertexCount));

  hash_table_.Resize(params);

  Reset();
}

Mesh::~Mesh() {
  checkCudaErrors(hipFree(mesh_data_.vertex_heap));
  checkCudaErrors(hipFree(mesh_data_.vertex_heap_counter));
  checkCudaErrors(hipFree(mesh_data_.vertices));
  checkCudaErrors(hipFree(mesh_data_.triangle_heap));
  checkCudaErrors(hipFree(mesh_data_.triangle_heap_counter));
  checkCudaErrors(hipFree(mesh_data_.triangles));
}

void Mesh::Reset() {
  const int threads_per_block = 64;
  const dim3 grid_size((kMaxVertexCount + threads_per_block - 1)
                       / threads_per_block, 1);
  const dim3 block_size(threads_per_block, 1);

  ResetHeapKernel<<<grid_size, block_size>>>(mesh_data_);
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipGetLastError());

  hash_table_.Reset();
}

/// Assume hash_table_ is compactified
void Mesh::MarchingCubes(Map *map) {
  uint occupied_block_count;
  checkCudaErrors(hipMemcpy(&occupied_block_count,
                             gpu_data().compacted_hash_entry_counter,
                             sizeof(uint), hipMemcpyDeviceToHost));
  if (occupied_block_count <= 0)
    return;

  const uint threads_per_block = BLOCK_SIZE;
  const dim3 grid_size(occupied_block_count, 1);
  const dim3 block_size(threads_per_block, 1);
  MarchingCubesKernel<<<grid_size, block_size>>>(map->gpu_data(), gpu_data(),
          mesh_data_);
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipGetLastError());

  GarbageCollectionKernel<<<grid_size, block_size>>>(gpu_data(), mesh_data_);
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipGetLastError());
}

void Mesh::SaveMesh(std::string path) {
  /// get data from GPU
  LOG(INFO) << "Copying data from GPU";
  Vertex* vertices = new Vertex[kMaxVertexCount];
  Triangle *triangles = new Triangle[kMaxVertexCount];
  checkCudaErrors(hipMemcpy(vertices, mesh_data_.vertices,
                             sizeof(Vertex) * kMaxVertexCount,
                             hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(triangles, mesh_data_.triangles,
                             sizeof(Triangle) * kMaxVertexCount,
                             hipMemcpyDeviceToHost));

  LOG(INFO) << "Writing data";
  std::ofstream out(path);
  std::stringstream ss;

  int vertex_count = 0;
  for (int i = 0; i < kMaxVertexCount; ++i) {
    ss.str("");
    ss <<  "v " << vertices[i].pos.x << " "
       << vertices[i].pos.y << " "
       << vertices[i].pos.z << "\n";
    //LOG(INFO) << ss.str();
    if (vertices[i].pos.x == 0.0f
            && vertices[i].pos.y == 0.0f
            && vertices[i].pos.z == 0.0f) continue;
    ++vertex_count;
    out << ss.str();
  }
  LOG(INFO) << "vertex count: " << vertex_count;

  int triangle_count = 0;
  for (int i = 0; i < kMaxVertexCount; ++i) {
    ss.str("");
    ss << "f " << triangles[i].vertex_ptrs.x + 1 << " "
       << triangles[i].vertex_ptrs.y + 1 << " "
       << triangles[i].vertex_ptrs.z + 1 << "\n";
    //LOG(INFO) << ss.str();
    if (triangles[i].vertex_ptrs.x == -1
        || triangles[i].vertex_ptrs.y == -1
        || triangles[i].vertex_ptrs.z == -1)
      continue;
    ++triangle_count;
    out << ss.str();
  }
  LOG(INFO) << "triangle count: " << triangle_count;

  delete[] vertices;
  delete[] triangles;
}