#include "hip/hip_runtime.h"
#include "mesh.h"

#include <hip/hip_runtime_api.h>
#include <>
#include <params.h>

////////////////////
/// class Mesh
////////////////////

////////////////////
/// Device code
////////////////////
__global__
void ResetHeapKernel(MeshGPU mesh,
                     int max_vertex_count,
                     int max_triangle_count) {
  const uint idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < max_vertex_count) {
    mesh.vertex_heap[idx] = max_vertex_count - idx - 1;
    mesh.vertices[idx].Clear();
  }
  if (idx < max_triangle_count) {
    mesh.triangle_heap[idx] = max_triangle_count - idx - 1;
    mesh.triangles[idx].Clear();
  }

}

////////////////////
/// Host code
////////////////////
Mesh::Mesh() {}

Mesh::~Mesh() {
  Free();
}

void Mesh::Alloc(const MeshParams &mesh_params) {
  checkCudaErrors(hipMalloc(&gpu_data_.vertex_heap,
                             sizeof(uint) * mesh_params.max_vertex_count));
  checkCudaErrors(hipMalloc(&gpu_data_.vertex_heap_counter, sizeof(uint)));
  checkCudaErrors(hipMalloc(&gpu_data_.vertices,
                             sizeof(Vertex) * mesh_params.max_vertex_count));

  checkCudaErrors(hipMalloc(&gpu_data_.triangle_heap,
                             sizeof(uint) * mesh_params.max_triangle_count));
  checkCudaErrors(hipMalloc(&gpu_data_.triangle_heap_counter, sizeof(uint)));
  checkCudaErrors(hipMalloc(&gpu_data_.triangles,
                             sizeof(Triangle) * mesh_params.max_triangle_count));
}

void Mesh::Free() {
  checkCudaErrors(hipFree(gpu_data_.vertex_heap));
  checkCudaErrors(hipFree(gpu_data_.vertex_heap_counter));
  checkCudaErrors(hipFree(gpu_data_.vertices));

  checkCudaErrors(hipFree(gpu_data_.triangle_heap));
  checkCudaErrors(hipFree(gpu_data_.triangle_heap_counter));
  checkCudaErrors(hipFree(gpu_data_.triangles));
}

void Mesh::Resize(const MeshParams &mesh_params) {
  mesh_params_ = mesh_params;
  Alloc(mesh_params);
  Reset();
}

void Mesh::Reset() {
  checkCudaErrors(hipMemcpy(gpu_data_.vertex_heap_counter,
                             &mesh_params_.max_vertex_count,
                             sizeof(uint),
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(gpu_data_.triangle_heap_counter,
                             &mesh_params_.max_triangle_count,
                             sizeof(uint),
                             hipMemcpyHostToDevice));

  const int threads_per_block = 64;
  const dim3 grid_size((kMaxVertexCount + threads_per_block - 1)
                       / threads_per_block, 1);
  const dim3 block_size(threads_per_block, 1);

  ResetHeapKernel<<<grid_size, block_size>>>(gpu_data_,
          mesh_params_.max_vertex_count,
          mesh_params_.max_triangle_count);
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipGetLastError());
}

////////////////////
/// class CompactMesh
////////////////////

/// Life cycle
CompactMesh::CompactMesh() {}

CompactMesh::~CompactMesh() {
  Free();
}

void CompactMesh::Alloc(const MeshParams &mesh_params) {
  checkCudaErrors(hipMalloc(&gpu_data_.vertex_index_remapper,
                             sizeof(int) * mesh_params.max_vertex_count));

  checkCudaErrors(hipMalloc(&gpu_data_.vertex_counter,
                             sizeof(uint)));
  checkCudaErrors(hipMalloc(&gpu_data_.vertices_ref_count,
                             sizeof(int) * mesh_params.max_vertex_count));
  checkCudaErrors(hipMalloc(&gpu_data_.vertices,
                             sizeof(float3) * mesh_params.max_vertex_count));
  checkCudaErrors(hipMalloc(&gpu_data_.normals,
                             sizeof(float3) * mesh_params.max_vertex_count));

  checkCudaErrors(hipMalloc(&gpu_data_.triangle_counter,
                             sizeof(uint)));
  checkCudaErrors(hipMalloc(&gpu_data_.triangles_ref_count,
                             sizeof(int) * mesh_params.max_triangle_count));
  checkCudaErrors(hipMalloc(&gpu_data_.triangles,
                             sizeof(int3) * mesh_params.max_triangle_count));
}

void CompactMesh::Free() {
  checkCudaErrors(hipFree(gpu_data_.vertex_index_remapper));

  checkCudaErrors(hipFree(gpu_data_.vertex_counter));
  checkCudaErrors(hipFree(gpu_data_.vertices_ref_count));
  checkCudaErrors(hipFree(gpu_data_.vertices));
  checkCudaErrors(hipFree(gpu_data_.normals));

  checkCudaErrors(hipFree(gpu_data_.triangle_counter));
  checkCudaErrors(hipFree(gpu_data_.triangles_ref_count));
  checkCudaErrors(hipFree(gpu_data_.triangles));
}

void CompactMesh::Resize(const MeshParams &mesh_params) {
  mesh_params_ = mesh_params;
  Alloc(mesh_params);
  Reset();
}

/// Reset
void CompactMesh::Reset() {
  checkCudaErrors(hipMemset(gpu_data_.vertex_index_remapper, 0xff,
                             sizeof(int) * mesh_params_.max_vertex_count));
  checkCudaErrors(hipMemset(gpu_data_.vertices_ref_count, 0,
                             sizeof(int) * mesh_params_.max_vertex_count));
  checkCudaErrors(hipMemset(gpu_data_.vertex_counter,
                             0, sizeof(uint)));
  checkCudaErrors(hipMemset(gpu_data_.triangles_ref_count, 0,
                             sizeof(int) * mesh_params_.max_triangle_count));
  checkCudaErrors(hipMemset(gpu_data_.triangle_counter,
                             0, sizeof(uint)));
}

uint CompactMesh::vertex_count() {
  uint compact_vertex_count;
  checkCudaErrors(hipMemcpy(&compact_vertex_count,
                             gpu_data_.vertex_counter,
                             sizeof(uint), hipMemcpyDeviceToHost));
  return compact_vertex_count;
}

uint CompactMesh::triangle_count() {
  uint compact_triangle_count;
  checkCudaErrors(hipMemcpy(&compact_triangle_count,
                             gpu_data_.triangle_counter,
                             sizeof(uint), hipMemcpyDeviceToHost));
  return compact_triangle_count;
}
