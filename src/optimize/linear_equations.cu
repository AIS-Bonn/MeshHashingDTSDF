#include "hip/hip_runtime.h"
//
// Created by wei on 17-10-25.
//

#include "linear_equations.h"

void SensorLinearEquations::Alloc(int width, int height) {
  width_ = width;
  height_ = height;

  if (!is_allocated_on_gpu_) {
    hipMalloc(&A, sizeof(float3x3) * width*height);
    hipMalloc(&b, sizeof(float3) * width*height);
    is_allocated_on_gpu_ = true;
  }
}

void SensorLinearEquations::Free() {
  if (is_allocated_on_gpu_) {
    hipFree(A);
    hipFree(b);
    is_allocated_on_gpu_ = false;
  }
}

void SensorLinearEquations::Reset() {
  if (is_allocated_on_gpu_) {
    hipMemset(A, 0, sizeof(float3x3)*width_*height_);
    hipMemset(b, 0, sizeof(float3)*width_*height_);
  }
}