#include "hip/hip_runtime.h"
/// 16 threads per block

#include "rgbd_sensor.h"
#include <extern/cuda/hip/hip_runtime_api.h>
#include <extern/cuda/hip/hip_runtime_api.h>
#include <extern/cuda/hip/hip_vector_types.h>
#include <geometry/geometry_helper.h>
#include <sensor/preprocess.h>
#include <util/debugging.hpp>
#include <visualization/color_util.h>

#include <hip/driver_types.h>
#include <glog/logging.h>


/// Member functions: (CPU code)
Sensor::Sensor(SensorParams &sensor_params) {
  const uint image_size = sensor_params.height * sensor_params.width;

  params_ = sensor_params; // Is it copy constructing?
  checkCudaErrors(hipMalloc(&data_.depth_buffer, sizeof(short) * image_size));
  checkCudaErrors(hipMalloc(&data_.color_buffer, sizeof(uchar4) * image_size));

  checkCudaErrors(hipMalloc(&data_.depth_data, sizeof(float) * image_size));
  checkCudaErrors(hipMalloc(&data_.inlier_ratio, sizeof(float) * image_size));
  checkCudaErrors(hipMalloc(&data_.filtered_depth_data, sizeof(float) * image_size));
  checkCudaErrors(hipMalloc(&data_.color_data, sizeof(float4) * image_size));
  checkCudaErrors(hipMalloc(&data_.normal_data, sizeof(float4) * image_size));

  data_.depth_channel_desc = hipCreateChannelDesc<float>();
  checkCudaErrors(hipMallocArray(&data_.depth_array,
                                  &data_.depth_channel_desc,
                                  params_.width, params_.height));

  data_.color_channel_desc = hipCreateChannelDesc<float4>();
  checkCudaErrors(hipMallocArray(&data_.color_array,
                                  &data_.color_channel_desc,
                                  params_.width, params_.height));

  data_.normal_channel_desc = hipCreateChannelDesc<float4>();
  checkCudaErrors(hipMallocArray(&data_.normal_array,
                                  &data_.normal_channel_desc,
                                  params_.width, params_.height));

  data_.depth_texture = 0;
  data_.color_texture = 0;
  data_.normal_texture = 0;

  BindCUDATexture();
  is_allocated_on_gpu_ = true;
}

Sensor::~Sensor() {
  if (is_allocated_on_gpu_) {
    checkCudaErrors(hipFree(data_.depth_buffer));
    checkCudaErrors(hipFree(data_.color_buffer));

    checkCudaErrors(hipFree(data_.depth_data));
    checkCudaErrors(hipFree(data_.inlier_ratio));
    checkCudaErrors(hipFree(data_.filtered_depth_data));
    checkCudaErrors(hipFree(data_.color_data));
    checkCudaErrors(hipFree(data_.normal_data));

    checkCudaErrors(hipFreeArray(data_.depth_array));
    checkCudaErrors(hipFreeArray(data_.color_array));
    checkCudaErrors(hipFreeArray(data_.normal_array));
  }
}

void Sensor::BindCUDATexture() {
  hipResourceDesc depth_resource;
  memset(&depth_resource, 0, sizeof(depth_resource));
  depth_resource.resType = hipResourceTypeArray;
  depth_resource.res.array.array = data_.depth_array;

  hipTextureDesc depth_tex_desc;
  memset(&depth_tex_desc, 0, sizeof(depth_tex_desc));
  depth_tex_desc.readMode = hipReadModeElementType;

  if (data_.depth_texture != 0)
    checkCudaErrors(hipDestroyTextureObject(data_.depth_texture));
  checkCudaErrors(hipCreateTextureObject(&data_.depth_texture,
                                          &depth_resource,
                                          &depth_tex_desc,
                                          NULL));

  hipResourceDesc color_resource;
  memset(&color_resource, 0, sizeof(color_resource));
  color_resource.resType = hipResourceTypeArray;
  color_resource.res.array.array = data_.color_array;

  hipTextureDesc color_tex_desc;
  memset(&color_tex_desc, 0, sizeof(color_tex_desc));
  color_tex_desc.readMode = hipReadModeElementType;

  if (data_.color_texture != 0)
    checkCudaErrors(hipDestroyTextureObject(data_.color_texture));
  checkCudaErrors(hipCreateTextureObject(&data_.color_texture,
                                          &color_resource,
                                          &color_tex_desc,
                                          NULL));

  hipResourceDesc normal_resource;
  memset(&normal_resource, 0, sizeof(normal_resource));
  normal_resource.resType = hipResourceTypeArray;
  normal_resource.res.array.array = data_.normal_array;

  hipTextureDesc normal_tex_desc;
  memset(&normal_tex_desc, 0, sizeof(normal_tex_desc));
  normal_tex_desc.readMode = hipReadModeElementType;

  if (data_.normal_texture != 0)
    checkCudaErrors(hipDestroyTextureObject(data_.normal_texture));
  checkCudaErrors(hipCreateTextureObject(&data_.normal_texture,
                                          &normal_resource,
                                          &normal_tex_desc,
                                          NULL));
}

int Sensor::Process(cv::Mat &depth, cv::Mat &color) {
  // TODO(wei): deal with distortion
  /// Disable all filters at current
  ConvertDepthFormat(depth, data_.depth_buffer, data_.depth_data, params_);
  ConvertColorFormat(color, data_.color_buffer, data_.color_data, params_);
  ResetInlierRatio(data_.inlier_ratio, params_);
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipGetLastError());

  ComputeNormalMap(data_.depth_data, data_.normal_data, params_);

  /// Array used as texture in mapper
  checkCudaErrors(hipMemcpyToArray(data_.depth_array, 0, 0,
                                    data_.depth_data,
                                    sizeof(float)*params_.height*params_.width,
                                    hipMemcpyDeviceToDevice));
  checkCudaErrors(hipMemcpyToArray(data_.color_array, 0, 0,
                                    data_.color_data,
                                    sizeof(float4)*params_.height*params_.width,
                                    hipMemcpyDeviceToDevice));

  checkCudaErrors(hipMemcpyToArray(data_.normal_array, 0, 0,
                                    data_.normal_data,
                                    sizeof(float4)*params_.height*params_.width,
                                    hipMemcpyDeviceToDevice));

  // Save debug normal image
//  static uint counter = 0;
//  std::stringstream ss;
//  ss << "/tmp/normals/normals" << std::setfill('0') << std::setw(4) << counter << ".png";
//  SaveNormalImage(ss.str(), data_, params_);
//  counter +=1;

  BindCUDATexture();
  return 0;
}
