#include "hip/hip_runtime.h"
/// 16 threads per block

#include "rgbd_sensor.h"
#include "geometry/geometry_helper.h"
#include "visualization/color_util.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include <glog/logging.h>
#include <hip/driver_types.h>
#include <extern/cuda/hip/hip_runtime_api.h>
#include "sensor/preprocess.h"


/// Member functions: (CPU code)
Sensor::Sensor(SensorParams &sensor_params) {
  const uint image_size = sensor_params.height * sensor_params.width;

  params_ = sensor_params; // Is it copy constructing?
  checkCudaErrors(hipMalloc(&data_.depth_buffer, sizeof(short) * image_size));
  checkCudaErrors(hipMalloc(&data_.color_buffer, sizeof(uchar4) * image_size));
  checkCudaErrors(hipMalloc(&data_.depth_data, sizeof(float) * image_size));
  checkCudaErrors(hipMalloc(&data_.color_data, sizeof(float4) * image_size));
  data_.depth_channel_desc = hipCreateChannelDesc<float>();
  checkCudaErrors(hipMallocArray(&data_.depth_array,
                                  &data_.depth_channel_desc,
                                  params_.width, params_.height));
  data_.color_channel_desc = hipCreateChannelDesc<float4>();
  checkCudaErrors(hipMallocArray(&data_.color_array,
                                  &data_.color_channel_desc,
                                  params_.width, params_.height));
  data_.depth_texture = 0;
  data_.color_texture = 0;

  BindCUDATexture();
  is_allocated_on_gpu_ = true;
}

Sensor::~Sensor() {
  if (is_allocated_on_gpu_) {
    checkCudaErrors(hipFree(data_.depth_buffer));
    checkCudaErrors(hipFree(data_.color_buffer));
    checkCudaErrors(hipFree(data_.depth_data));
    checkCudaErrors(hipFree(data_.color_data));
    checkCudaErrors(hipFreeArray(data_.depth_array));
    checkCudaErrors(hipFreeArray(data_.color_array));
  }
}

void Sensor::BindCUDATexture() {
  hipResourceDesc depth_resource;
  memset(&depth_resource, 0, sizeof(depth_resource));
  depth_resource.resType = hipResourceTypeArray;
  depth_resource.res.array.array = data_.depth_array;

  hipTextureDesc depth_tex_desc;
  memset(&depth_tex_desc, 0, sizeof(depth_tex_desc));
  depth_tex_desc.readMode = hipReadModeElementType;

  if (data_.depth_texture != 0)
    checkCudaErrors(hipDestroyTextureObject(data_.depth_texture));
  checkCudaErrors(hipCreateTextureObject(&data_.depth_texture,
                                          &depth_resource,
                                          &depth_tex_desc,
                                          NULL));

  hipResourceDesc color_resource;
  memset(&color_resource, 0, sizeof(color_resource));
  color_resource.resType = hipResourceTypeArray;
  color_resource.res.array.array = data_.color_array;

  hipTextureDesc color_tex_desc;
  memset(&color_tex_desc, 0, sizeof(color_tex_desc));
  color_tex_desc.readMode = hipReadModeElementType;

  if (data_.color_texture != 0)
    checkCudaErrors(hipDestroyTextureObject(data_.color_texture));
  checkCudaErrors(hipCreateTextureObject(&data_.color_texture, &color_resource, &color_tex_desc, NULL));
}

int Sensor::Process(cv::Mat &depth, cv::Mat &color) {
  // TODO(wei): deal with distortion
  /// Disable all filters at current

  ConvertDepthFormat(depth, data_.depth_buffer, data_.depth_data, params_);
  ConvertColorFormat(color, data_.color_buffer, data_.color_data, params_);

  /// Array used as texture in mapper
  checkCudaErrors(hipMemcpyToArray(data_.depth_array, 0, 0,
                                    data_.depth_data,
                                    sizeof(float)*params_.height*params_.width,
                                    hipMemcpyDeviceToDevice));
  checkCudaErrors(hipMemcpyToArray(data_.color_array, 0, 0,
                                    data_.color_data,
                                    sizeof(float4)*params_.height*params_.width,
                                    hipMemcpyDeviceToDevice));
  BindCUDATexture();
  return 0;
}



